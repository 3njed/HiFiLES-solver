#include "hip/hip_runtime.h"
/*!
 * \file cuda_kernels.cu
 * \brief _____________________________
 * \author - Original code: SD++ developed by Patrice Castonguay, Antony Jameson,
 *                          Peter Vincent, David Williams (alphabetical by surname).
 *         - Current development: Aerospace Computing Laboratory (ACL) directed
 *                                by Prof. Jameson. (Aero/Astro Dept. Stanford University).
 * \version 1.0.0
 *
 * HiFiLES (High Fidelity Large Eddy Simulation).
 * Copyright (C) 2013 Aerospace Computing Laboratory.
 */

#define HALFWARP 16
#include <iostream>

using namespace std;

#include "../include/cuda_kernels.h"
#include "../include/error.h"
#include "../include/util.h"

#ifdef _MPI
#include "mpi.h"
#endif

//Key

// met[0][0] = rx
// met[1][0] = sx
// met[0][1] = ry
// met[1][1] = sy

// Add a bespoke_MV_kernel to do non-sparse matrix-vector multiplication

template<int n_fields> 
__global__ void bespoke_SPMV_kernel(double *g_c, double *g_b, double *g_cont_mat, int *g_col_mat, const int n_nz, const int n_cells, const int dim1, const int dim0, const int cells_per_block, const int stride_n, const int stride_m, int add_flag)
{
	extern __shared__ double s_b[];

	const int tid = threadIdx.x;
	const int ic_loc = tid/dim0;
	const int ifp = tid-ic_loc*dim0;
	const int ic = blockIdx.x*cells_per_block+ ic_loc;
	const int stride_shared = cells_per_block*dim1+ (cells_per_block*dim1)/HALFWARP;
	int i_qpt, m, m1;

  double q[n_fields];

  #pragma unroll
  for (int j=0;j<n_fields;j++)
    q[j] = 0.;

	double mat_entry;

  	if (tid < cells_per_block*dim0 && ic < n_cells)
  	{

		// Fetching data to shared memory
  		int n_fetch_loops = (dim1-1)/(dim0)+1;

		// Since n_qpts might be larger than dim0 
        	// each thread might have to fetch more than n_fields values
	  	for (int i=0;i<n_fetch_loops;i++)
  		{
     			i_qpt= i*dim0+ifp;

     			if (i_qpt<dim1)
     		 	{
				    // Fetch the four field values of solution point i_qpt
				    m  = ic_loc *dim1+i_qpt;
				    m += m/HALFWARP;

				    m1 = ic     *dim1+i_qpt;
            #pragma unroll  
            for (int j=0;j<n_fields;j++)
            {
				      s_b[m] = g_b[m1]; // copy global B vector to shared B vector
              m += stride_shared; 
              m1 += stride_n;
            }
			    }
     	} 
   	}

   	__syncthreads(); // make sure memory copy is complete in all threads

   	if (tid < cells_per_block*dim0 && ic < n_cells)
   	{

  		// With data in shared memory, perform matrix multiplication
  		// 1 thread per flux point
  		for (int i=0;i<n_nz;i++)
  		{
			  m = i*dim0+ifp;
			  m1 = dim1*ic_loc + g_col_mat[m];
			  //m1 = n_qpts*ic_loc + tex1Dfetch(t_col_mat,m);
			  m1 += m1/HALFWARP;

			  mat_entry = g_cont_mat[m];
			  //mat_entry = fetch_double(t_cont_mat,m);

        #pragma unroll
        for (int j=0;j<n_fields;j++)
        {
			    q[j] += mat_entry*s_b[m1];	
          m1 += stride_shared;
        }

  		}
	
		// Store in global memory
		 m = ic*dim0+ifp;
    #pragma unroll
     for (int j=0;j<n_fields;j++)
      {
       if (add_flag==0)
        g_c[m] = q[j];
       else if (add_flag==1)
        g_c[m] += q[j];

       m += stride_m;
     }
  	} 
	
}


template<int n_dims, int n_fields> 
__device__ void set_inv_boundary_conditions_kernel(int bdy_type, double* u_l, double* u_r, double* norm, double* loc, double *bdy_params, double gamma, double R_ref, double time_bound, int equation)
{
  double p_l, p_r;
  double T_l, T_r;
  double v_sq,vn_l;
  double rho_bound = bdy_params[0];
  double* v_bound = &bdy_params[1];
  double p_bound = bdy_params[4];
  double* v_wall = &bdy_params[5];
  double T_wall = bdy_params[8];

  double x,y,z;

  x = loc[0];
  y = loc[1];

  if(n_dims==3)
    z = loc[2];

  //printf("x = %6.10f\n",x);
  //printf("y = %6.10f\n",y);

  if(equation==0) //Navier-Stokes BC's
  {
    // Compute pressure on left side
    v_sq = 0.;
    for (int i=0;i<n_dims;i++)
      v_sq += (u_l[i+1]*u_l[i+1]);
	  p_l   = (gamma-1.0)*( u_l[n_dims+1] - 0.5*v_sq/u_l[0]);

    // Compute normal velocity on left side
    vn_l = 0.;
    for (int i=0;i<n_dims;i++)
      vn_l += u_l[i+1]*norm[i];
    vn_l /= u_l[0];

	  if(bdy_type == 1)
	  // subsonic inflow simple (free pressure)
	  {
	  	// fix density and velocity
	  	u_r[0] =  rho_bound;
      for (int i=0;i<n_dims;i++)
        u_r[i+1] = v_bound[i];

	  	// extrapolate pressure
	  	p_r   = p_l; 

	  	// compute energy
      v_sq = 0.;
      for (int i=0;i<n_dims;i++)
        v_sq += (u_r[i+1]*u_r[i+1]);
	    u_r[n_dims+1] = (p_r/(gamma-1.0)) + 0.5*v_sq/u_r[0];
	  } 

	  // subsonic outflow simple (fixed pressure)
	  else if(bdy_type == 2)
	  {
	  	// extrapolate density and velocity
	  	u_r[0]    = u_l[0];
      for (int i=0;i<n_dims;i++)
        u_r[i+1] = u_l[i+1];

	  	// fix pressure
	  	p_r = p_bound;

	  	// compute energy
      v_sq = 0.;
      for (int i=0;i<n_dims;i++)
        v_sq += (u_r[i+1]*u_r[i+1]);
	    u_r[n_dims+1] = (p_r/(gamma-1.0)) + 0.5*v_sq/u_r[0];
	  
	  }

	  // subsonic inflow characteristic
	  else if(bdy_type == 3)
	  {
	  	// TODO: Implement characteristic subsonic inflow BC
	  	printf("subsonic inflow char not implemented in 3d");
	  }
	  //subsonic outflow characteristic
	  else if(bdy_type == 4)
	  {
	  	printf("subsonic outflow char not implemented in 3d");
	  }

	  // supersonic inflow
	  else if(bdy_type == 5)
	  {
	  	// fix density and velocity
	  	u_r[0] =  rho_bound;
      for (int i=0;i<n_dims;i++)
        u_r[i+1] = v_bound[i];

	  	// fix pressure
	  	p_r = p_bound;	

	  	// compute energy
      v_sq = 0.;
      for (int i=0;i<n_dims;i++)
        v_sq += (u_r[i+1]*u_r[i+1]);
	    u_r[n_dims+1] = (p_r/(gamma-1.0)) + 0.5*v_sq/u_r[0];


	  }

	  // supersonic outflow
	  else if(bdy_type == 6)
	  {
	  	// extrapolate density, velocity
	  	u_r[0] = u_l[0];
      for (int i=0;i<n_dims;i++)
        u_r[i+1] = u_l[i+1];

	  	// pressure and energy
      u_r[n_dims+1]=u_l[n_dims+1];
	  }

	  // slip wall
	  else if(bdy_type == 7)
	  {
	  	// extrapolate density
	  	u_r[0] = u_l[0];

	  	// reflect normal momentum
      for (int i=0;i<n_dims;i++)
	  	  u_r[i+1] = u_l[i+1]-2.0*vn_l*u_l[0]*norm[i];

	  	// extrapolate energy
	  	u_r[n_dims+1] = u_l[n_dims+1];
	  }
	  
    // isothermal, no-slip wall (fixed)
    else if(bdy_type == 11)
	  {
	  	// extrapolate pressure
	  	p_r = p_l; 
      //p_r = p_bound; //HACK

      // isothermal temperature
      T_r = T_wall;
      //T_l = p_l/(u_l[0]*R_ref);

      // density
      u_r[0] = p_r/(R_ref*T_r);
      
      // no-slip
      for (int i=0;i<n_dims;i++)
        u_r[i+1] = 0.;

	  	// energy
      v_sq = 0.;
      for (int i=0;i<n_dims;i++)
        v_sq += (u_r[i+1]*u_r[i+1]);
	    u_r[n_dims+1] = (p_r/(gamma-1.0)) + 0.5*v_sq/u_r[0];
	  } 

    // adiabatic, no-slip wall (fixed)
    else if(bdy_type == 12)
    {
      // extrapolate density    
      u_r[0] = u_l[0];

      // extrapolate pressure
	  	p_r = p_l; 
      
      // no-slip
      for (int i=0;i<n_dims;i++)
        u_r[i+1] = 0.;
	  	
      // energy
      v_sq = 0.;
      for (int i=0;i<n_dims;i++)
        v_sq += (u_r[i+1]*u_r[i+1]);
	    u_r[n_dims+1] = (p_r/(gamma-1.0)) + 0.5*v_sq/u_r[0];
    }

    // isothermal, no-slip wall (moving)
    else if(bdy_type == 13)
    {
      // extrapolate pressure
	  	p_r = p_l; 
      //p_r = p_bound; //HACK
      
      // isothermal temperature
      T_r = T_wall;
      //T_l = p_l/(u_l[0]*R_ref);

      // density
      u_r[0] = p_r/(R_ref*T_r);
      
      // no-slip
      for (int i=0;i<n_dims;i++)
        u_r[i+1] = u_r[0]*v_wall[i];
      
	  	// energy
      v_sq = 0.;
      for (int i=0;i<n_dims;i++)
        v_sq += (u_r[i+1]*u_r[i+1]);
	    u_r[n_dims+1] = (p_r/(gamma-1.0)) + 0.5*v_sq/u_r[0];
    }

    // adiabatic, no-slip wall (moving)
    else if(bdy_type == 14)
    {
      // extrapolate density    
      u_r[0] = u_l[0];

      // extrapolate pressure
	  	p_r = p_l; 
      
      // no-slip
      for (int i=0;i<n_dims;i++)
        u_r[i+1] = u_r[0]*v_wall[i];
	  	
      // energy
      v_sq = 0.;
      for (int i=0;i<n_dims;i++)
        v_sq += (u_r[i+1]*u_r[i+1]);
	    u_r[n_dims+1] = (p_r/(gamma-1.0)) + 0.5*v_sq/u_r[0];
    }

	  else if (bdy_type == 15) // Characteristic
	  {
	  	double c_star;
	  	double vn_star;
      double vn_bound;
      double vt_star;
	  	double r_plus,r_minus;
	  	
	  	double one_over_s;
	  	double h_free_stream;

      vn_bound = 0;
      for (int i=0;i<n_dims;i++)
        vn_bound += v_bound[i]*norm[i];

	  	r_plus  = vn_l + 2./(gamma-1.)*sqrt(gamma*p_l/u_l[0]);
	  	r_minus = vn_bound - 2./(gamma-1.)*sqrt(gamma*p_bound/rho_bound);

	  	c_star = 0.25*(gamma-1.)*(r_plus-r_minus);
	  	vn_star = 0.5*(r_plus+r_minus);
 
	  	//Works only for 2D and quasi-2D

	  	if (vn_l<0) // Inflow
	  	{
	  		vt_star = (v_bound[0]*norm[1] - v_bound[1]*norm[0]); 
	  		//assumes quasi-2D boundary i.e. norm[2] == 0;

        // HACK
	  		one_over_s = pow(rho_bound,gamma)/p_bound;

	  		// freestream total enthalpy
        v_sq = 0.;
        for (int i=0;i<n_dims;i++)
          v_sq += v_bound[i]*v_bound[i];

	  		h_free_stream = gamma/(gamma-1.)*p_bound/rho_bound+ 0.5*v_sq;
	  		u_r[0] = pow(1./gamma*(one_over_s*c_star*c_star),1./(gamma-1.));

      	u_r[1] = u_r[0]*(norm[0]*vn_star + norm[1]*vt_star);
        u_r[2] = u_r[0]*(norm[1]*vn_star - norm[0]*vt_star);
	  		
	  		if(n_dims==3)
	  		{
	  			u_r[3] = 0.0; //no cross-flow
	  		}

	  		p_r = u_r[0]/gamma*c_star*c_star;
	  		u_r[n_dims+1] = u_r[0]*h_free_stream - p_r;
	  	}
	  	else  // Outflow
	  	{
	  		vt_star = (u_l[1]*norm[1] - u_l[2]*norm[0])/u_l[0];

	  		one_over_s = pow( u_l[0], gamma)/p_l;				

	  		// freestream total enthalpy
	  		u_r[0] = pow( (1./gamma*(one_over_s*c_star*c_star)) , (1./(gamma-1.)));

        u_r[1] = u_r[0]*(norm[0]*vn_star + norm[1]*vt_star);
        u_r[2] = u_r[0]*(norm[1]*vn_star - norm[0]*vt_star);

	  		// no cross-flow
	  		if(n_dims==3)
	  		{
	  			u_r[3] = 0.0;	
	  		}

	  		p_r = u_r[0]/gamma*c_star*c_star;
        v_sq = 0.;
        for (int i=0;i<n_dims;i++)
          v_sq += u_r[i+1]*u_r[i+1];

	  		u_r[n_dims+1] = (p_r/(gamma-1.0)) + 0.5*v_sq/u_r[0];;
	  	}

	  }
    else if (bdy_type==16) // Dual consistent BC
    {


      // "DUAL-CONSISTENT" WALL BC

      u_r[0]   = u_l[0];
      u_r[1]  = (1-norm[0]*norm[0])*u_l[1] - norm[0]*norm[1]*u_l[2];
      u_r[2]  = (1-norm[1]*norm[1])*u_l[2] - norm[0]*norm[1]*u_l[1];
      u_r[3]  = u_l[3];


    }
	  else if (bdy_type == 17) // Characteristic Lala
	  {

      if (n_dims==3)
        printf("Char BDY does not work in 3D");

      double RHOL,UL,VL,PL,VNL,VTL,CL,SL;
      double VNI,VTI,CI,SI,RM,RP,VNB,VTB,SB,CB,RHOB,UB,VB,PB;

      RHOL  = u_l[0];
      UL    = u_l[1]/RHOL;
      VL    = u_l[2]/RHOL;
      PL    = (gamma-1) * (u_l[3]-0.5*RHOL*(UL*UL+VL*VL));
      VNL   = UL*norm[0] + VL*norm[1];
      VTL   = VL*norm[0] - UL*norm[1];
      CL    = sqrt(gamma*PL/RHOL);
      SL    = PL/pow(RHOL,gamma);

      //printf("v_bound= %f %f, v_l = %f %f\n",v_bound[0],v_bound[1],UL,VL);
      //printf("p_bound= %f,  p_l = %f\n",p_bound,PL);
      //printf("rho_bound= %f,  rho_l = %f\n",rho_bound,RHOL);

      // FLOW PROPERTIES IN THE FARFIELD
      VNI   = v_bound[0]*norm[0] + v_bound[1]*norm[1];
      VTI   = v_bound[1]*norm[0] - v_bound[0]*norm[1];
      CI    = sqrt(gamma*p_bound/rho_bound);
      SI    = p_bound/pow(rho_bound,gamma);

      // CALCULATE THE RIEMANN INVARIANT
      RM    = VNI - 2./(gamma-1.)*CI;
      RP    = VNL + 2./(gamma-1.)*CL;
      VNB   = (RP + RM)/2.;

      //printf("norm= %f %f, vnb=%f \n",norm[0],norm[1],VNB);

      if( VNB <0 ) { // INFLOW
         VTB      = VTI;
         SB       = SI;
      }
      else {// OUTFLOW
         VTB      = VTL;
         SB       = SL;
      }

      CB    = 0.25*(gamma-1.)*(RP - RM);

      RHOB = pow(CB*CB/(gamma*SB),1./(gamma-1.));
      PB    = SB*pow(RHOB,gamma);
      UB    = norm[0]*VNB - norm[1]*VTB;
      VB    = norm[1]*VNB + norm[0]*VTB;

      u_r[0] = RHOB;
      u_r[1] = RHOB*UB;
      u_r[2] = RHOB*VB;
      u_r[3] = PB/(gamma-1.) + 0.5*RHOB*(UB*UB + VB*VB);

	  }  





	  else
	  {
	  	// Boundary condition not implemented yet
	  	printf("bdy_type=%d\n",bdy_type);
	  	printf("Boundary conditions yet to be implemented");
	  }
  }


  if(equation==1) //Advection/Advection-Diffusion BC's
  {
    if(bdy_type==50) //Trivial dirichlet
    {
      u_r[0]=0.0; 
    }
  }

}


template<int n_dims, int n_fields>
__device__ void set_vis_boundary_conditions_kernel(int bdy_type, double* u_l, double* u_r, double* grad_u, double *norm, double *loc, double *bdy_params, double gamma, double R_ref, double time_bound, int equation)
{
  int cpu_flag;
  cpu_flag = 0;
  
    double v_sq;
  double inte;
  double p_l, p_r;
  
  double grad_vel[n_dims*n_dims];


// Adiabatic wall
if(bdy_type == 12 || bdy_type == 14) 
{
  v_sq = 0.;
  for (int i=0;i<n_dims;i++)
    v_sq += (u_l[i+1]*u_l[i+1]);
	p_l   = (gamma-1.0)*( u_l[n_dims+1] - 0.5*v_sq/u_l[0]);
  p_r = p_l;
  
  inte = p_r/((gamma-1.0)*u_r[0]);
  
  if(cpu_flag)
  {
    // Velocity gradients
    for (int j=0;j<n_dims;j++) 
    {
      for (int i=0;i<n_dims;i++) 
        grad_vel[j*n_dims + i] = (grad_u[i*n_fields + (j+1)] - grad_u[i*n_fields + 0]*u_r[j+1]/u_r[0])/u_r[0];
    }
    
    // Energy gradients (grad T = 0)
    if(n_dims == 2)
    {
      for (int i=0;i<n_dims;i++) 
        grad_u[i*n_fields + 3] = inte*grad_u[i*n_fields + 0] + 0.5*((u_r[1]*u_r[1]+u_r[2]*u_r[2])/(u_r[0]*u_r[0]))*grad_u[i*n_fields + 0] + u_r[0]*((u_r[1]/u_r[0])*grad_vel[0*n_dims + i]+(u_r[2]/u_r[0])*grad_vel[1*n_dims + i]); 
    }
    else if(n_dims == 3)
    {
      for (int i=0;i<n_dims;i++) 
        grad_u[i*n_fields + 4] = inte*grad_u[i*n_fields + 0] + 0.5*((u_r[1]*u_r[1]+u_r[2]*u_r[2]+u_r[3]*u_r[3])/(u_r[0]*u_r[0]))*grad_u[i*n_fields + 0] + u_r[0]*((u_r[1]/u_r[0])*grad_vel[0*n_dims + i]+(u_r[2]/u_r[0])*grad_vel[1*n_dims + i]+(u_r[3]/u_r[0])*grad_vel[2*n_dims + i]); 
    }
  }
  else
  {
    // Velocity gradients
    for (int j=0;j<n_dims;j++) 
    {
      for (int i=0;i<n_dims;i++) 
        grad_vel[j*n_dims + i] = (grad_u[(j+1)*n_dims + i] - grad_u[0*n_dims + i]*u_r[j+1]/u_r[0])/u_r[0];
    }
    
    if(n_dims == 2)
    {
      // Total energy gradient
      for (int i=0;i<n_dims;i++) 
        grad_u[3*n_dims + i] = inte*grad_u[0*n_dims + i] + 0.5*((u_r[1]*u_r[1]+u_r[2]*u_r[2])/(u_r[0]*u_r[0]))*grad_u[0*n_dims + i] + u_r[0]*((u_r[1]/u_r[0])*grad_vel[0*n_dims + i]+(u_r[2]/u_r[0])*grad_vel[1*n_dims + i]); 
    }
    else if(n_dims == 3)
    {
      for (int i=0;i<n_dims;i++) 
        grad_u[4*n_dims + i] = inte*grad_u[0*n_dims + i] + 0.5*((u_r[1]*u_r[1]+u_r[2]*u_r[2]+u_r[3]*u_r[3])/(u_r[0]*u_r[0]))*grad_u[0*n_dims + i] + u_r[0]*((u_r[1]/u_r[0])*grad_vel[0*n_dims + i]+(u_r[2]/u_r[0])*grad_vel[1*n_dims + i]+(u_r[3]/u_r[0])*grad_vel[2*n_dims + i]); 
    }
  }

}   

}


template<int in_n_dims>
__device__ void inv_NS_flux(double* q, double *p, double* f, double in_gamma, int in_field)
{
  if(in_n_dims==2) {
    
    if (in_field==-1) {
		  (*p) = (in_gamma-1.0)*(q[3]-0.5*(q[1]*q[1]+q[2]*q[2])/q[0]);
    }
    else if (in_field==0) {
		  f[0] = q[1];
		  f[1] = q[2];
    }
    else if (in_field==1) {
		  f[0]  = (*p)+(q[1]*q[1]/q[0]);
		  f[1]  = q[2]*q[1]/q[0];
    }
    else if (in_field==2) {
		  f[0]  = q[1]*q[2]/q[0];
		  f[1]  = (*p) + (q[2]*q[2]/q[0]);
    }
    else if (in_field==3) {
		  f[0]  = q[1]/q[0]*(q[3]+(*p));
		  f[1]  = q[2]/q[0]*(q[3]+(*p));
    }
  }
  else if(in_n_dims==3)
  {
    if (in_field==-1) {
			(*p) = (in_gamma-1.0)*(q[4]-0.5*(q[1]*q[1]+q[2]*q[2]+q[3]*q[3])/q[0]);
    }
    else if (in_field==0) {
			f[0] = q[1];
			f[1] = q[2];
			f[2] = q[3]; 
    }
    else if (in_field==1) {
	    f[0] = (*p)+(q[1]*q[1]/q[0]);
			f[1] = q[2]*q[1]/q[0];
			f[2] = q[3]*q[1]/q[0];
    }
    else if (in_field==2) {
			f[0] = q[1]*q[2]/q[0];
			f[1] = (*p)+(q[2]*q[2]/q[0]);
			f[2] = q[3]*q[2]/q[0];
    }
    else if (in_field==3) {
			f[0] = q[1]*q[3]/q[0];
			f[1] = q[2]*q[3]/q[0];
			f[2] = (*p) + (q[3]*q[3]/q[0]);
    }
    else if (in_field==4) {
	 		f[0] = q[1]/q[0]*(q[4]+(*p));
			f[1] = q[2]/q[0]*(q[4]+(*p));
			f[2] = q[3]/q[0]*(q[4]+(*p));
    }
  }
}


template<int in_n_dims>
__device__ void vis_NS_flux(double* q, double* grad_q, double* grad_vel, double* grad_ene, double* stensor, double* f, double* inte, double* mu, double in_prandtl, double in_gamma, double in_rt_inf, double in_mu_inf, double in_c_sth, double in_fix_vis, int in_field)
{
  double diag;
  double rt_ratio;

  if(in_n_dims==2) {
      
    if(in_field==-1) {
    
      // Internal energy
      (*inte) = (q[3]/q[0])-0.5*((q[1]*q[1]+q[2]*q[2])/(q[0]*q[0]));
    
      // Viscosity
      rt_ratio = (in_gamma-1.)*(*inte)/(in_rt_inf);
      (*mu) = in_mu_inf*pow(rt_ratio,1.5)*(1.+in_c_sth)/(rt_ratio+in_c_sth);
      (*mu) = (*mu) + in_fix_vis*(in_mu_inf - (*mu));
      
      // Velocity gradients
      #pragma unroll
      for (int j=0;j<in_n_dims;j++) 
      {
        #pragma unroll
        for (int i=0;i<in_n_dims;i++) 
          grad_vel[j*in_n_dims + i] = (grad_q[(j+1)*in_n_dims + i] - grad_q[0*in_n_dims + i]*q[j+1]/q[0])/q[0];
      }
      
      // Kinetic energy gradient
      #pragma unroll
      for (int i=0;i<in_n_dims;i++) 
        grad_ene[i] = 0.5*((q[1]*q[1]+q[2]*q[2])/(q[0]*q[0]))*grad_q[0*in_n_dims + i] + q[0]*((q[1]/q[0])*grad_vel[0*in_n_dims + i]+(q[2]/q[0])*grad_vel[1*in_n_dims + i]); 

      // Total energy gradient
      #pragma unroll
      for (int i=0;i<in_n_dims;i++) 
        grad_ene[i] = (grad_q[3*in_n_dims + i] - grad_ene[i] - grad_q[0*in_n_dims + i]*(*inte))/q[0];

      diag = (grad_vel[0*in_n_dims + 0] + grad_vel[1*in_n_dims + 1])/3.0;

      // Stress tensor
      #pragma unroll
      for (int i=0;i<in_n_dims;i++) 
        stensor[i] = 2.0*(*mu)*(grad_vel[i*in_n_dims + i] - diag);
      
      stensor[2] = (*mu)*(grad_vel[0*in_n_dims + 1] + grad_vel[1*in_n_dims + 0]); 

    }
    else if (in_field==0) {
      f[0] = 0.0;
      f[1] = 0.0;
    }
    else if (in_field==1) {
		  f[0]  = -stensor[0];
		  f[1]  = -stensor[2];
    }
    else if (in_field==2) {
		  f[0]  = -stensor[2];
		  f[1]  = -stensor[1];
    }
    else if (in_field==3) {
		  f[0]  = -((q[1]/q[0])*stensor[0] + (q[2]/q[0])*stensor[2] + (*mu)*in_gamma*grad_ene[0]/in_prandtl);
		  f[1]  = -((q[1]/q[0])*stensor[2] + (q[2]/q[0])*stensor[1] + (*mu)*in_gamma*grad_ene[1]/in_prandtl);
    }
  }
  else if(in_n_dims==3)
  {
    if(in_field==-1) {
      
      // Internal energy
      (*inte) = (q[4]/q[0])-0.5*((q[1]*q[1]+q[2]*q[2]+q[3]*q[3])/(q[0]*q[0]));
    
      // Viscosity
      rt_ratio = (in_gamma-1.)*(*inte)/(in_rt_inf);
      (*mu) = in_mu_inf*pow(rt_ratio,1.5)*(1.+in_c_sth)/(rt_ratio+in_c_sth);
      (*mu) = (*mu) + in_fix_vis*(in_mu_inf - (*mu));
      
      // Velocity gradients
      #pragma unroll
      for (int j=0;j<in_n_dims;j++) 
      {
        #pragma unroll
        for (int i=0;i<in_n_dims;i++) 
          grad_vel[j*in_n_dims + i] = (grad_q[(j+1)*in_n_dims + i] - grad_q[0*in_n_dims + i]*q[j+1]/q[0])/q[0];
      }
      
      // Kinetic energy gradient
      #pragma unroll
      for (int i=0;i<in_n_dims;i++) 
        grad_ene[i] = 0.5*((q[1]*q[1]+q[2]*q[2]+q[3]*q[3])/(q[0]*q[0]))*grad_q[0*in_n_dims + i] + q[0]*((q[1]/q[0])*grad_vel[0*in_n_dims + i]+(q[2]/q[0])*grad_vel[1*in_n_dims + i]+(q[3]/q[0])*grad_vel[2*in_n_dims + i]); 

      // Total energy gradient
      #pragma unroll
      for (int i=0;i<in_n_dims;i++) 
        grad_ene[i] = (grad_q[4*in_n_dims + i] - grad_ene[i] - grad_q[0*in_n_dims + i]*(*inte))/q[0];

      diag = (grad_vel[0*in_n_dims + 0] + grad_vel[1*in_n_dims + 1] + grad_vel[2*in_n_dims + 2])/3.0;

      // Stress tensor
      #pragma unroll
      for (int i=0;i<in_n_dims;i++) 
        stensor[i] = 2.0*(*mu)*(grad_vel[i*in_n_dims + i] - diag);
      
      stensor[3] = (*mu)*(grad_vel[0*in_n_dims + 1] + grad_vel[1*in_n_dims + 0]); 
      stensor[4] = (*mu)*(grad_vel[0*in_n_dims + 2] + grad_vel[2*in_n_dims + 0]); 
      stensor[5] = (*mu)*(grad_vel[1*in_n_dims + 2] + grad_vel[2*in_n_dims + 1]); 
    }
    else if (in_field==0) {
      f[0] = 0.0;
			f[1] = 0.0;
			f[2] = 0.0; 
    }
    else if (in_field==1) {
      f[0]  = -stensor[0];
			f[1]  = -stensor[3];
			f[2]  = -stensor[4];
    }
    else if (in_field==2) {
			f[0] = -stensor[3];
			f[1] = -stensor[1];
			f[2] = -stensor[5];
    }
    else if (in_field==3) {
			f[0] = -stensor[4];
			f[1] = -stensor[5];
			f[2] = -stensor[2];
    }
    else if (in_field==4) {
	 		f[0] = -((q[1]/q[0])*stensor[0]+(q[2]/q[0])*stensor[3]+(q[3]/q[0])*stensor[4] + (*mu)*in_gamma*grad_ene[0]/in_prandtl);
			f[1] = -((q[1]/q[0])*stensor[3]+(q[2]/q[0])*stensor[1]+(q[3]/q[0])*stensor[5] + (*mu)*in_gamma*grad_ene[1]/in_prandtl);
			f[2] = -((q[1]/q[0])*stensor[4]+(q[2]/q[0])*stensor[5]+(q[3]/q[0])*stensor[2] + (*mu)*in_gamma*grad_ene[2]/in_prandtl);
    }
  }
}

__device__ double SGS_filter_width(double in_detjac, int in_ele_type, int in_n_dims, double in_filter_ratio)
{
	// Define filter width by Deardorff's unstructured element method
	double delta, vol;

	if (in_ele_type==0) // triangle
	{
		vol = in_detjac*2.0;
	}
	else if (in_ele_type==1) // quads
	{
		vol = in_detjac*4.0;
	}
	else if (in_ele_type==2) // tets
	{
		vol = in_detjac*8.0/6.0;
	}
	else if (in_ele_type==4) // hexas
	{
		vol = in_detjac*8.0;
	}

	delta = in_filter_ratio*pow(vol,1./in_n_dims);
	//printf("vol: %6.10f",vol);
	//printf("delta: %6.10f",delta);
	return delta;
}

template<int in_n_dims>
__device__ void SGS_flux_kernel(double* q, double* grad_q, double* grad_vel, double* grad_ene, double* sdtensor, double* straintensor, double* Leonard_mom, double* Leonard_ene, double* f, int SGS_model, double delta, double in_gamma, int in_field)
{
	int i, j;
	int eddy, sim;
	double Cs, mu_t;
	double Smod=0.0;
	double Pr=0.5; // turbulent Prandtl number
	double num=0.0;
	double denom=0.0;
	double diag=0.0;
	double eps=1.e-10;

	// Set flags depending on which SGS model we are using
	// 0: Smagorinsky, 1: WALE, 2: WALE-similarity, 3: SVV, 4: Similarity
	if(SGS_model==0) {
		eddy = 1;
		sim = 0;
	}
	else if(SGS_model==1) {
		eddy = 1;
		sim = 0;
	}
	else if(SGS_model==2) {
		eddy = 1;
		sim = 1;
	}
	else if(SGS_model==3) {
		eddy = 0;
		sim = 0;
	}
	else if(SGS_model==4) {
		eddy = 0;
		sim = 1;
	}

	// Calculate eddy viscosity

	// Smagorinsky model
	if(SGS_model==0) {

		Cs=0.1;

		// Calculate modulus of strain rate tensor
		#pragma unroll
		for (i=0;i<in_n_dims;i++) {
			Smod += 2.0*straintensor[i]*straintensor[i];
		}

		// Now the off-diagonal components of strain tensor:
	  if(in_n_dims==2) {
			Smod += 4.0*straintensor[2]*straintensor[2];
		}
	  else if(in_n_dims==3) {
			Smod += 4.0*(straintensor[3]*straintensor[3] + straintensor[4]*straintensor[4] + straintensor[5]*straintensor[5]);
		}

		// Finally, the modulus of strain rate tensor
		Smod = sqrt(Smod);

		mu_t = q[0]*Cs*Cs*delta*delta*Smod;
	}

	// WALE or WSM model
	else if(SGS_model==1 || SGS_model==2) {

		Cs=0.5;

		// Square of velocity gradient tensor
		#pragma unroll
		for (i=0;i<in_n_dims;i++) {
			sdtensor[i] = 0.0;
			#pragma unroll
			for (j=0;j<in_n_dims;j++) {
				diag += grad_vel[i*in_n_dims + j]*grad_vel[j*in_n_dims + i]/3.0;
				sdtensor[i] += grad_vel[i*in_n_dims + j]*grad_vel[j*in_n_dims + i];
			}
		}

		// subtract trace from diagonal entries of tensor
		#pragma unroll
		for (i=0;i<in_n_dims;i++)
			sdtensor[i] -= diag;

		// off-diagonal terms of tensor
	  if(in_n_dims==2) {
			sdtensor[2] = 0.0;
			#pragma unroll
			for (j=0;j<in_n_dims;j++) {
				sdtensor[2] += (grad_vel[0*in_n_dims + j]*grad_vel[j*in_n_dims + 1] + grad_vel[1*in_n_dims + j]*grad_vel[j*in_n_dims + 0])/2.0;
			}
		}
	  else if(in_n_dims==3) {
			sdtensor[3] = 0.0;
			sdtensor[4] = 0.0;
			sdtensor[5] = 0.0;
			#pragma unroll
			for (j=0;j<in_n_dims;j++) {
				sdtensor[3] += (grad_vel[0*in_n_dims + j]*grad_vel[j*in_n_dims + 1] + grad_vel[1*in_n_dims + j]*grad_vel[j*in_n_dims + 0])/2.0;

				sdtensor[4] += (grad_vel[0*in_n_dims + j]*grad_vel[j*in_n_dims + 2] + grad_vel[2*in_n_dims + j]*grad_vel[j*in_n_dims + 0])/2.0;

				sdtensor[5] += (grad_vel[1*in_n_dims + j]*grad_vel[j*in_n_dims + 2] + grad_vel[2*in_n_dims + j]*grad_vel[j*in_n_dims + 1])/2.0;
			}
		}

		// numerator and denominator of eddy viscosity term
		#pragma unroll
		for (i=0;i<in_n_dims;i++) {
			num += sdtensor[i]*sdtensor[i];
			denom += straintensor[i]*straintensor[i];
		}

		if(in_n_dims==2) {
			num += 2.0*sdtensor[2]*sdtensor[2];
			denom += 2.0*straintensor[2]*straintensor[2];
		}
		else if(in_n_dims==3) {
			num += 2.0*(sdtensor[3]*sdtensor[3] + sdtensor[4]*sdtensor[4] + sdtensor[5]*sdtensor[5]);
			denom += 2.0*(straintensor[3]*straintensor[3] + straintensor[4]*straintensor[4] + straintensor[5]*straintensor[5]);
		}

		denom = pow(denom,2.5) + pow(num,1.25);
		num = pow(num,1.5);
		mu_t = q[0]*Cs*Cs*delta*delta*num/(denom+eps);
	}

	// Now set the flux values
	if (eddy==1) {
		if (in_n_dims==2) {

			// Density
			if (in_field==0) {
				f[0] = 0.0;
				f[1] = 0.0;
			}
			// u
			else if (in_field==1) {
				f[0] = -2.0*mu_t*straintensor[0];
				f[1] = -2.0*mu_t*straintensor[2];
			}
			// v
			else if (in_field==2) {
				f[0] = -2.0*mu_t*straintensor[2];
				f[1] = -2.0*mu_t*straintensor[1];
			}
			// Energy
			else if (in_field==3) {
				f[0] = -1.0*in_gamma*mu_t/Pr*grad_ene[0];
				f[1] = -1.0*in_gamma*mu_t/Pr*grad_ene[1];
			}
		}
		else if(in_n_dims==3) {

			// Density
			if (in_field==0) {
				f[0] = 0.0;
				f[1] = 0.0;
				f[2] = 0.0; 
			}
			// u
			else if (in_field==1) {
				f[0] = -2.0*mu_t*straintensor[0];
				f[1] = -2.0*mu_t*straintensor[3];
				f[2] = -2.0*mu_t*straintensor[4];
			}
			// v
			else if (in_field==2) {
				f[0] = -2.0*mu_t*straintensor[3];
				f[1] = -2.0*mu_t*straintensor[1];
				f[2] = -2.0*mu_t*straintensor[5];
			}
			// w
			else if (in_field==3) {
				f[0] = -2.0*mu_t*straintensor[4];
				f[1] = -2.0*mu_t*straintensor[5];
				f[2] = -2.0*mu_t*straintensor[2];
			}
			// Energy
			else if (in_field==4) {
				f[0] = -1.0*in_gamma*mu_t/Pr*grad_ene[0];
				f[1] = -1.0*in_gamma*mu_t/Pr*grad_ene[1];
				f[2] = -1.0*in_gamma*mu_t/Pr*grad_ene[2];
			}
		}
	}
	// Add similarity term to SGS fluxes if WSM or Similarity model
	if (sim==1)
	{
		if(in_n_dims==2) {

			// Density
			if (in_field==0) {
				f[0] += 0.0;
				f[1] += 0.0;
			}

			// u
			if (in_field==1) {
				f[0] += q[0]*Leonard_mom[0];
				f[1] += q[0]*Leonard_mom[2];
			}
			// v
			else if (in_field==2) {
				f[0] += q[0]*Leonard_mom[2];
				f[1] += q[0]*Leonard_mom[1];
			}
			// Energy
			else if (in_field==3) {
				f[0] += q[0]*in_gamma*Leonard_ene[0];
				f[1] += q[0]*in_gamma*Leonard_ene[1];
			}
		}
		else if(in_n_dims==3)
		{
			// u
			if (in_field==1) {
				f[0] += q[0]*Leonard_mom[0];
				f[1] += q[0]*Leonard_mom[3];
				f[2] += q[0]*Leonard_mom[4];
			}
			// v
			else if (in_field==2) {
				f[0] += q[0]*Leonard_mom[3];
				f[1] += q[0]*Leonard_mom[1];
				f[2] += q[0]*Leonard_mom[5];
			}
			// w
			else if (in_field==3) {
				f[0] += q[0]*Leonard_mom[4];
				f[1] += q[0]*Leonard_mom[5];
				f[2] += q[0]*Leonard_mom[2];
			}
			// Energy
			else if (in_field==4) {
				f[0] += q[0]*in_gamma*Leonard_ene[0];
				f[1] += q[0]*in_gamma*Leonard_ene[1];
				f[2] += q[0]*in_gamma*Leonard_ene[2];
			}
		}
	}
}

template<int in_n_fields, int in_n_dims>
__device__ __host__ void rusanov_flux(double* q_l, double *q_r, double *norm, double *fn, double in_gamma)
{
  double vn_l, vn_r;
  double vn_av_mag, c_av;
  double p_l, p_r,f_l,f_r;
  
  double f[in_n_dims];

  // Compute normal velocity
  vn_l = 0.;
  vn_r = 0.;
  #pragma unroll
  for (int i=0;i<in_n_dims;i++) {
    vn_l += q_l[i+1]/q_l[0]*norm[i];
    vn_r += q_r[i+1]/q_r[0]*norm[i];
  }

  // Flux prep
  inv_NS_flux<in_n_dims>(q_l,&p_l,f,in_gamma,-1);
  inv_NS_flux<in_n_dims>(q_r,&p_r,f,in_gamma,-1);
	  		
	vn_av_mag=sqrt(0.25*(vn_l+vn_r)*(vn_l+vn_r));
	c_av=sqrt((in_gamma*(p_l+p_r))/(q_l[0]+q_r[0]));

  #pragma unroll
  for (int i=0;i<in_n_fields;i++)
  {
    // Left normal flux
    inv_NS_flux<in_n_dims>(q_l,&p_l,f,in_gamma,i);
    
    f_l = f[0]*norm[0] + f[1]*norm[1];
    if(in_n_dims==3)
      f_l += f[2]*norm[2];
      
    // Right normal flux
    inv_NS_flux<in_n_dims>(q_r,&p_r,f,in_gamma,i);
    
    f_r = f[0]*norm[0] + f[1]*norm[1];
    if(in_n_dims==3)
      f_r += f[2]*norm[2];
    
    // Common normal flux
    fn[i] = 0.5*(f_l+f_r) - 0.5*(vn_av_mag+c_av)*(q_r[i]-q_l[i]);
  }
}


template<int in_n_fields, int in_n_dims>
__device__ __host__ void right_flux(double *q_r, double *norm, double *fn, double in_gamma)
{

  double p_r,f_r;
  double f[in_n_dims];

  // Flux prep
  inv_NS_flux<in_n_dims>(q_r,&p_r,f,in_gamma,-1);

  #pragma unroll
  for (int i=0;i<in_n_fields;i++)
  {
    //Right normal flux
    inv_NS_flux<in_n_dims>(q_r,&p_r,f,in_gamma,i);
    
    f_r = f[0]*norm[0] + f[1]*norm[1];
    if(in_n_dims==3)
      f_r += f[2]*norm[2];
    
    fn[i] = f_r;
  }
}


template<int n_fields, int n_dims>
__device__ __host__ void roe_flux(double* u_l, double *u_r, double *norm, double *fn, double in_gamma)
{
	double p_l,p_r;
  double h_l, h_r;
  double sq_rho,rrho,hm,usq,am,am_sq,unm;
  double lambda0,lambdaP,lambdaM;
  double rhoun_l, rhoun_r,eps;
  double a1,a2,a3,a4,a5,a6,aL1,bL1;
  double v_l[n_dims],v_r[n_dims],um[n_dims],du[n_fields];
  //array<double> um(n_dims);

  // velocities
  #pragma unroll
  for (int i=0;i<n_dims;i++)  {
    v_l[i] = u_l[i+1]/u_l[0];
    v_r[i] = u_r[i+1]/u_r[0];
  }

  if (n_dims==2) {
	  p_l=(in_gamma-1.0)*(u_l[3]-(0.5*u_l[0]*((v_l[0]*v_l[0])+(v_l[1]*v_l[1]))));
	  p_r=(in_gamma-1.0)*(u_r[3]-(0.5*u_r[0]*((v_r[0]*v_r[0])+(v_r[1]*v_r[1]))));
  }
  else
    printf("Roe not implemented in 3D\n");

  h_l = (u_l[n_dims+1]+p_l)/u_l[0];
  h_r = (u_r[n_dims+1]+p_r)/u_r[0];

  sq_rho = sqrt(u_r[0]/u_l[0]);

  rrho = 1./(sq_rho+1.);

  #pragma unroll
  for (int i=0;i<n_dims;i++)
    um[i] = rrho*(v_l[i]+sq_rho*v_r[i]);

  hm      = rrho*(h_l     +sq_rho*h_r);

  //if (flag)
  //  printf("hm = %16.12f, um=%16.12f %16.12f %16.12f\n",hm,um[0],um[1],um[2]);

  usq=0.;
  #pragma unroll
  for (int i=0;i<n_dims;i++)
    usq += 0.5*um[i]*um[i];

  am_sq   = (in_gamma-1.)*(hm-usq);
  am  = sqrt(am_sq);

  unm = 0.;
  #pragma unroll
  for (int i=0;i<n_dims;i++)
    unm += um[i]*norm[i];

  //if (flag)
  //  printf("unm=%16.12f, usq=%16.12f\n",unm,usq);

  // Compute Euler flux (first part)
  rhoun_l = 0.;
  rhoun_r = 0.;

  #pragma unroll
  for (int i=0;i<n_dims;i++)
  {
    rhoun_l += u_l[i+1]*norm[i];
    rhoun_r += u_r[i+1]*norm[i];
  }

  if (n_dims==2)
  {
    fn[0] = rhoun_l + rhoun_r;
    fn[1] = rhoun_l*v_l[0] + rhoun_r*v_r[0] + (p_l+p_r)*norm[0];
    fn[2] = rhoun_l*v_l[1] + rhoun_r*v_r[1] + (p_l+p_r)*norm[1];
    fn[3] = rhoun_l*h_l   +rhoun_r*h_r;

    //if (flag)
    //  printf("fn=%16.12f %16.12f %16.12f %16.12f\n",fn[0],fn[1],fn[2],fn[3]);
  }
  else
    printf("Roe not implemented in 3D\n");

  #pragma unroll
  for (int i=0;i<n_fields;i++)
  {
    du[i] = u_r[i]-u_l[i];
    //if (flag)
    //  printf("du=%16.12f\n",du[i]);
  }

  lambda0 = abs(unm);
  lambdaP = abs(unm+am);
  lambdaM = abs(unm-am);

  // Entropy fix
  eps = 0.5*(abs(rhoun_l/u_l[0]-rhoun_r/u_r[0])+ abs(sqrt(in_gamma*p_l/u_l[0])-sqrt(in_gamma*p_r/u_r[0])));
  if(lambda0 < 2.*eps) 
    lambda0 = 0.25*lambda0*lambda0/eps + eps;
  if(lambdaP < 2.*eps) 
    lambdaP = 0.25*lambdaP*lambdaP/eps + eps;
  if(lambdaM < 2.*eps) 
    lambdaM = 0.25*lambdaM*lambdaM/eps + eps;


  a2 = 0.5*(lambdaP+lambdaM)-lambda0;
  a3 = 0.5*(lambdaP-lambdaM)/am;
  a1 = a2*(in_gamma-1.)/am_sq;
  a4 = a3*(in_gamma-1.);

    //if (flag)
    //  printf("ndims=%d\n",n_dims);

  if (n_dims==2)
  {
    //if (flag)
    //  printf("inside");

    //if (flag)
    //  printf("%16.12f %16.12f %16.12f %16.12f %16.12f %16.12f %16.12f %16.12f\n",usq,du[0],um[0],du[1],um[1],du[2],du[3]);


    a5 = usq*du[0]-um[0]*du[1]-um[1]*du[2]+du[3];
    a6 = unm*du[0]-norm[0]*du[1]-norm[1]*du[2];
  }
  else if (n_dims==3)
  {
    a5 = usq*du[0]-um[0]*du[1]-um[1]*du[2]-um[2]*du[3]+du[4];
    a6 = unm*du[0]-norm[0]*du[1]-norm[1]*du[2]-norm[2]*du[3];
  }

  //if (flag)
  // printf("a=%16.12f %16.12f %16.12f %16.12f %16.12f %16.12f \n",a2,a3,a1,a4,a5,a6);

  aL1 = a1*a5 - a3*a6;
  bL1 = a4*a5 - a2*a6;

  //if (flag)
  // printf("aL1=%16.12f %16.12f \n",aL1,bL1);

  // Compute Euler flux (second part)
  if (n_dims==2)
  {
    fn[0] = fn[0] - (lambda0*du[0]+aL1);
    fn[1] = fn[1] - (lambda0*du[1]+aL1*um[0]+bL1*norm[0]);
    fn[2] = fn[2] - (lambda0*du[2]+aL1*um[1]+bL1*norm[1]);
    fn[3] = fn[3] - (lambda0*du[3]+aL1*hm   +bL1*unm);

    //if (flag)
    //  printf("fn=%16.12f %16.12f %16.12f %16.12f\n",fn[0],fn[1],fn[2],fn[3]);
  }
  else if (n_dims==3)
  {
    fn[0] = fn[0] - (lambda0*du[0]+aL1);
    fn[1] = fn[1] - (lambda0*du[1]+aL1*um[0]+bL1*norm[0]);
    fn[2] = fn[2] - (lambda0*du[2]+aL1*um[1]+bL1*norm[1]);
    fn[3] = fn[3] - (lambda0*du[3]+aL1*um[2]+bL1*norm[2]);
    fn[4] = fn[4] - (lambda0*du[4]+aL1*hm   +bL1*unm);
  }

  #pragma unroll
  for (int i=0;i<n_fields;i++)
    fn[i] =  0.5*fn[i];

}


template<int n_dims>
__device__ __host__ void lax_friedrichs_flux(double* u_l, double *u_r, double *norm, double *fn, double wave_speed_x, double wave_speed_y, double wave_speed_z, double lambda)
{
		double u_av, u_diff;
		double norm_speed;
    
		u_av = 0.5*(u_r[0]+u_l[0]);
    u_diff = u_l[0]-u_r[0]; 

    norm_speed=0.;
    if (n_dims==2)
      norm_speed += wave_speed_x*norm[0] + wave_speed_y*norm[1];
    else if (n_dims==3)
      norm_speed += wave_speed_x*norm[0] + wave_speed_y*norm[1] + wave_speed_z*norm[2];
	  		
    // Compute common interface flux
    fn[0] = 0.;
    if (n_dims==2)
      fn[0] += (wave_speed_x*norm[0] + wave_speed_y*norm[1])*u_av;
    else if (n_dims==3)
      fn[0] += (wave_speed_x*norm[0] + wave_speed_y*norm[1] + wave_speed_z*norm[2])*u_av;
    fn[0] += 0.5*lambda*abs(norm_speed)*u_diff;
}


template<int in_n_dims, int n_fields, int flux_spec>
__device__ void ldg_solution(double* q_l, double* q_r, double* norm, double* q_c, double in_pen_fact)
{
  if(flux_spec==0) // Interior, mpi
  {
    // Choosing a unique direction for the switch

		if(in_n_dims==2)
		{
    	if ((norm[0]+norm[1]) < 0.)
      	in_pen_fact=-in_pen_fact;
		}
		if(in_n_dims==3)
		{
    	if ((norm[0]+norm[1]+sqrt(2.)*norm[2]) < 0.)
      	in_pen_fact=-in_pen_fact;
		}

    #pragma unroll
    for (int i=0;i<n_fields;i++) 
      q_c[i] = 0.5*(q_l[i]+q_r[i]) - in_pen_fact*(q_l[i]-q_r[i]);
  }
  else if(flux_spec==1) // Dirichlet
  {
    #pragma unroll
    for (int i=0;i<n_fields;i++) 
      q_c[i] = q_r[i];
  }
  else if(flux_spec==2) // von Neumann
  {
    #pragma unroll
    for (int i=0;i<n_fields;i++) 
      q_c[i] = q_l[i];
  }
}


template<int in_n_dims, int in_flux_spec>
__device__ __host__ void ldg_flux(double q_l, double q_r, double* f_l, double* f_r, double* f_c, double* norm, double in_pen_fact, double in_tau)
{
  if(in_flux_spec==0) //Interior, mpi
  {
		if(in_n_dims==2)
		{
    	if ((norm[0]+norm[1]) < 0.)
      	in_pen_fact=-in_pen_fact;
		}
		if(in_n_dims==3)
		{
    	if ((norm[0]+norm[1]+sqrt(2.)*norm[2]) < 0.)
      	in_pen_fact=-in_pen_fact;
		}

    // Compute common interface flux
    #pragma unroll
    for (int i=0;i<in_n_dims;i++)
    {
      f_c[i] = 0.5*(f_l[i] + f_r[i]) + in_tau*norm[i]*(q_l - q_r);
      #pragma unroll
      for (int k=0;k<in_n_dims;k++)
        f_c[i] += in_pen_fact*norm[i]*norm[k]*(f_l[k] - f_r[k]);
    }
  }
  else if(in_flux_spec==1) // Dirichlet
  {
    #pragma unroll
    for (int i=0;i<in_n_dims;i++)
      f_c[i] = f_l[i] + in_tau*norm[i]*(q_l - q_r);
  }
  else if(in_flux_spec==2) // von Neumann
  {
    #pragma unroll
    for (int i=0;i<in_n_dims;i++)
      f_c[i] = f_r[i];
  }
}


template< int n_fields >
__global__ void RK11_update_kernel(double *g_q_qpts, double *g_div_tfg_con_qpts, double *g_jac_det_qpts, 
								const int n_cells, const int n_qpts, const double dt, const double const_src_term)
{
  int n = blockIdx.x*blockDim.x + threadIdx.x;
	const int m = n; 
	double jac;
  int stride = n_cells*n_qpts;
	 
	if (n<n_cells*n_qpts)
	{
		jac = g_jac_det_qpts[m];
		// Update 5 fields
    #pragma unroll
		for (int i=0;i<n_fields;i++)
		{
	  	g_q_qpts[n] -= dt*(g_div_tfg_con_qpts[n]/jac - const_src_term);
			n += stride;
		}
	}
}


template< int n_fields >
__global__ void RK45_update_kernel(double *g_q_qpts, double *g_div_tfg_con_qpts, double *g_res_qpts, double *g_jac_det_qpts, 
								const int n_cells, const int n_qpts, const double fa, const double fb, const double dt, const double const_src_term)
{
  int n = blockIdx.x*blockDim.x + threadIdx.x;
	const int m = n; 
	double rhs,res,jac;
  int stride = n_cells*n_qpts;
	 
	if (n<n_cells*n_qpts)
	{
		jac = g_jac_det_qpts[m];
		// Update 5 fields
    #pragma unroll
		for (int i=0;i<n_fields;i++)
		{
			rhs = -(g_div_tfg_con_qpts[n]/jac - const_src_term);
	  	res = g_res_qpts[n];
	  	res = fa*res + dt*rhs;
	 		g_res_qpts[n] = res;
	  	g_q_qpts[n] += fb*res;
			n += stride;
		}
	}
}


// gpu kernel to calculate transformed discontinuous inviscid flux at solution points for the wave equation
// otherwise, switch to one thread per output?
template<int in_n_dims>
__global__ void calc_tdisinvf_upts_AD_gpu_kernel(int in_n_upts_per_ele, int in_n_eles, double* in_disu_upts_ptr, double* out_tdisf_upts_ptr, double* in_detjac_upts_ptr, double* in_inv_detjac_mul_jac_upts_ptr, double wave_speed_x, double wave_speed_y, double wave_speed_z)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;

  double q;
  double f[in_n_dims];	
  double met[in_n_dims][in_n_dims];

	int stride = in_n_upts_per_ele*in_n_eles;

 	if(thread_id<(in_n_upts_per_ele*in_n_eles))
 	{
    q = in_disu_upts_ptr[thread_id];

    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
      #pragma unroll
      for (int j=0;j<in_n_dims;j++) 
        met[j][i] = in_inv_detjac_mul_jac_upts_ptr[thread_id + (i*in_n_dims+j)*stride];

    int index;

    if (in_n_dims==2)
    {
		  f[0] = wave_speed_x*q;
		  f[1] = wave_speed_y*q;

		  index = thread_id;
		  out_tdisf_upts_ptr[index       ] = met[0][0]*f[0] + met[0][1]*f[1]; 
		  out_tdisf_upts_ptr[index+stride] = met[1][0]*f[0] + met[1][1]*f[1];
    }
    else if (in_n_dims==3)
    {
			f[0] = wave_speed_x*q;
			f[1] = wave_speed_y*q;
			f[2] = wave_speed_z*q;

		  index = thread_id;
			out_tdisf_upts_ptr[index          ] = met[0][0]*f[0] + met[0][1]*f[1] + met[0][2]*f[2]; 
			out_tdisf_upts_ptr[index+  stride ] = met[1][0]*f[0] + met[1][1]*f[1] + met[1][2]*f[2]; 
			out_tdisf_upts_ptr[index+2*stride ] = met[2][0]*f[0] + met[2][1]*f[1] + met[2][2]*f[2]; 

    }
  }
}
/*! gpu kernel to calculate velocity and energy product terms for similarity model */
template<int in_n_fields>
__global__ void calc_similarity_terms_kernel(int in_n_upts_per_ele, int in_n_eles, int in_n_dims, double* in_disu_upts_ptr, double* in_uu_ptr, double* in_ue_ptr)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;

	int stride = in_n_upts_per_ele*in_n_eles;
	int i;
  double q[in_n_fields];
	double rsq;

 	if(thread_id<in_n_upts_per_ele*in_n_eles) {

    // Solution
    #pragma unroll
    for (i=0;i<in_n_fields;i++) {
      q[i] = in_disu_upts_ptr[thread_id + i*stride];
		}

		rsq = q[0]*q[0];

		if(in_n_dims==2) {
			/*! velocity-velocity product */
			in_uu_ptr[thread_id + 0*stride] = q[1]*q[1]/rsq;
			in_uu_ptr[thread_id + 1*stride] = q[2]*q[2]/rsq;
			in_uu_ptr[thread_id + 2*stride] = q[1]*q[2]/rsq;

			/*! velocity-energy product */
			q[3] -= 0.5*(q[1]*q[1] + q[2]*q[2])/q[0]; // internal energy*rho

			in_ue_ptr[thread_id + 0*stride] = q[1]*q[3]/rsq; // subtract kinetic energy
			in_ue_ptr[thread_id + 1*stride] = q[2]*q[3]/rsq;
		}
		else if(in_n_dims==3) {
			/*! velocity-velocity product */
			in_uu_ptr[thread_id + 0*stride] = q[1]*q[1]/rsq;
			in_uu_ptr[thread_id + 1*stride] = q[2]*q[2]/rsq;
			in_uu_ptr[thread_id + 2*stride] = q[3]*q[3]/rsq;
			in_uu_ptr[thread_id + 3*stride] = q[1]*q[2]/rsq;
			in_uu_ptr[thread_id + 4*stride] = q[1]*q[3]/rsq;
			in_uu_ptr[thread_id + 5*stride] = q[2]*q[3]/rsq;

			/*! velocity-energy product */
			q[4] -= 0.5*(q[1]*q[1] + q[2]*q[2] + q[3]*q[3])/q[0]; // internal energy*rho

			in_ue_ptr[thread_id + 0*stride] = q[1]*q[4]/rsq; // subtract kinetic energy
			in_ue_ptr[thread_id + 1*stride] = q[2]*q[4]/rsq;
			in_ue_ptr[thread_id + 2*stride] = q[3]*q[4]/rsq;
		}
	}
}

/*! gpu kernel to calculate Leonard tensors for similarity model */
template<int in_n_fields>
__global__ void calc_similarity_model_kernel(int in_n_upts_per_ele, int in_n_eles, int in_n_dims, double* in_disuf_upts_ptr, double* in_Lu_ptr, double* in_Le_ptr)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;

	int stride = in_n_upts_per_ele*in_n_eles;
	int i;
  double q[in_n_fields];
	double diag, rsq;

 	if(thread_id<in_n_upts_per_ele*in_n_eles) {
	  // filtered solution
  	#pragma unroll
  	for (i=0;i<in_n_fields;i++) {
  	  q[i] = in_disuf_upts_ptr[thread_id + i*stride];
		}

		rsq = q[0]*q[0];

		/*! subtract product of filtered solution terms from Leonard tensors */
		if(in_n_dims==2) {
			in_Lu_ptr[thread_id + 0*stride] = (in_Lu_ptr[thread_id + 0*stride] - q[1]*q[1])/rsq;
			in_Lu_ptr[thread_id + 1*stride] = (in_Lu_ptr[thread_id + 1*stride] - q[2]*q[2])/rsq;
			in_Lu_ptr[thread_id + 2*stride] = (in_Lu_ptr[thread_id + 2*stride] - q[1]*q[2])/rsq;

			diag = (in_Lu_ptr[thread_id + 0*stride] + in_Lu_ptr[thread_id + 1*stride])/3.0;

			q[3] -= 0.5*(q[1]*q[1] + q[2]*q[2])/q[0]; // internal energy*rho

			in_Le_ptr[thread_id + 0*stride] = (in_Le_ptr[thread_id + 0*stride] - q[1]*q[3])/rsq;
			in_Le_ptr[thread_id + 1*stride] = (in_Le_ptr[thread_id + 1*stride] - q[2]*q[3])/rsq;
		}
		else if(in_n_dims==3) {
			in_Lu_ptr[thread_id + 0*stride] = (in_Lu_ptr[thread_id + 0*stride] - q[1]*q[1])/rsq;
  	  in_Lu_ptr[thread_id + 1*stride] = (in_Lu_ptr[thread_id + 1*stride] - q[2]*q[2])/rsq;
  	  in_Lu_ptr[thread_id + 2*stride] = (in_Lu_ptr[thread_id + 2*stride] - q[3]*q[3])/rsq;
			in_Lu_ptr[thread_id + 3*stride] = (in_Lu_ptr[thread_id + 3*stride] - q[1]*q[2])/rsq;
  	  in_Lu_ptr[thread_id + 4*stride] = (in_Lu_ptr[thread_id + 4*stride] - q[1]*q[3])/rsq;
  	  in_Lu_ptr[thread_id + 5*stride] = (in_Lu_ptr[thread_id + 5*stride] - q[2]*q[3])/rsq;

			diag = (in_Lu_ptr[thread_id + 0*stride] + in_Lu_ptr[thread_id + 1*stride] + in_Lu_ptr[thread_id + 2*stride])/3.0;

			q[4] -= 0.5*(q[1]*q[1] + q[2]*q[2] + q[3]*q[3])/q[0]; // internal energy*rho

  	  in_Le_ptr[thread_id + 0*stride] = (in_Le_ptr[thread_id + 0*stride] - q[1]*q[4])/rsq;
  	  in_Le_ptr[thread_id + 1*stride] = (in_Le_ptr[thread_id + 1*stride] - q[2]*q[4])/rsq;
  	  in_Le_ptr[thread_id + 2*stride] = (in_Le_ptr[thread_id + 2*stride] - q[3]*q[4])/rsq;
		}

		/*! subtract diagonal from Lu */
  	#pragma unroll
		for (i=0;i<in_n_dims;++i) {
			in_Lu_ptr[thread_id + i*stride] -= diag;
		}
		// subtract diagonal from Le?
	}
}

/*! wrapper for gpu kernel to calculate terms for similarity model */
void calc_similarity_model_kernel_wrapper(int in_n_fields, int in_n_upts_per_ele, int in_n_eles, int in_n_dims, double* in_disu_upts_ptr, double* in_disuf_upts_ptr, double* in_uu_ptr, double* in_ue_ptr, double* in_Lu_ptr, double* in_Le_ptr, double* in_filter_upts_ptr, int flag)
{
  check_cuda_error("Before", __FILE__, __LINE__);

	// HACK: fix 256 threads per block
	int n_blocks=((in_n_eles*in_n_upts_per_ele-1)/256)+1;

	/*! Calculate product terms uu, ue */
	if (flag==0) {
		// fixed n_fields at 4 for 2d and 5 for 3d
		if(in_n_dims==2) {
			calc_similarity_terms_kernel <4> <<< n_blocks,256>>> (in_n_upts_per_ele, in_n_eles, in_n_dims, in_disu_upts_ptr, in_uu_ptr, in_ue_ptr);
		}
		else if(in_n_dims==3) {
			calc_similarity_terms_kernel <5> <<< n_blocks,256>>> (in_n_upts_per_ele, in_n_eles, in_n_dims, in_disu_upts_ptr, in_uu_ptr, in_ue_ptr);
		}
	}
	/*! Calculate Leonard tensors Lu, Le */
	else if (flag==1) {
		// fixed n_fields at 4 for 2d and 5 for 3d
		if(in_n_dims==2) {
			calc_similarity_model_kernel <4> <<< n_blocks,256>>> (in_n_upts_per_ele, in_n_eles, in_n_dims, in_disuf_upts_ptr, in_Lu_ptr, in_Le_ptr);
		}
		else if(in_n_dims==3) {
			calc_similarity_model_kernel <5> <<< n_blocks,256>>> (in_n_upts_per_ele, in_n_eles, in_n_dims, in_disuf_upts_ptr, in_Lu_ptr, in_Le_ptr);
		}
	}

  check_cuda_error("After",__FILE__, __LINE__);
}

// gpu kernel to calculate transformed discontinuous inviscid flux at solution points for the Navier-Stokes equation
// otherwise, switch to one thread per output?
template<int in_n_dims, int in_n_fields>
__global__ void calc_tdisinvf_upts_NS_gpu_kernel(int in_n_upts_per_ele, int in_n_eles, double* in_disu_upts_ptr, double* out_tdisf_upts_ptr, double* in_detjac_upts_ptr, double* in_inv_detjac_mul_jac_upts_ptr, double in_gamma)
{

	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;

  double q[in_n_fields];
  double f[in_n_dims];	
  double met[in_n_dims][in_n_dims];

	double p;
	int stride = in_n_upts_per_ele*in_n_eles;

 	if(thread_id<(in_n_upts_per_ele*in_n_eles))
 	{
    // Solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
      q[i] = in_disu_upts_ptr[thread_id + i*stride];

    // Metric terms
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
      #pragma unroll
      for (int j=0;j<in_n_dims;j++) 
        met[j][i] = in_inv_detjac_mul_jac_upts_ptr[thread_id + (i*in_n_dims+j)*stride];

    // Flux prep
    inv_NS_flux<in_n_dims>(q,&p,f,in_gamma,-1);

    int index;

    // Flux computation
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
    {
      inv_NS_flux<in_n_dims>(q,&p,f,in_gamma,i);
      
      index = thread_id+i*stride;
    
      if (in_n_dims==2) {
		    out_tdisf_upts_ptr[index                    ] = met[0][0]*f[0] + met[0][1]*f[1]; 
		    out_tdisf_upts_ptr[index+stride*in_n_fields ] = met[1][0]*f[0] + met[1][1]*f[1];
      }
      else if(in_n_dims==3)
      {
			  out_tdisf_upts_ptr[index                      ] = met[0][0]*f[0] + met[0][1]*f[1] + met[0][2]*f[2]; 
			  out_tdisf_upts_ptr[index+  stride*in_n_fields ] = met[1][0]*f[0] + met[1][1]*f[1] + met[1][2]*f[2]; 
			  out_tdisf_upts_ptr[index+2*stride*in_n_fields ] = met[2][0]*f[0] + met[2][1]*f[1] + met[2][2]*f[2]; 
      }
    }
  
  }
}


// gpu kernel to calculate normal transformed continuous inviscid flux at the flux points
template <int in_n_dims, int in_n_fields, int in_riemann_solve_type, int in_vis_riemann_solve_type>
__global__ void calc_norm_tconinvf_fpts_NS_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_norm_tconf_fpts_r_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr, double** in_norm_fpts_ptr, double** in_delta_disu_fpts_l_ptr, double** in_delta_disu_fpts_r_ptr, double in_gamma, double in_pen_fact, int in_viscous)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int stride = in_n_fpts_per_inter*in_n_inters;
	
  double q_l[in_n_fields]; 
  double q_r[in_n_fields]; 
  double fn[in_n_fields];
  double norm[in_n_dims];
  
  double q_c[in_n_fields];

  double jac;

	if(thread_id<stride)
  {  
    // Compute left state solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_l[i]=(*(in_disu_fpts_l_ptr[thread_id+i*stride])); 

    // Compute right state solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_r[i]=(*(in_disu_fpts_r_ptr[thread_id+i*stride])); 

    // Compute normal
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  norm[i]=*(in_norm_fpts_ptr[thread_id + i*stride]);

    if (in_riemann_solve_type==0)
      rusanov_flux<in_n_fields,in_n_dims> (q_l,q_r,norm,fn,in_gamma);
    else if (in_riemann_solve_type==2)
      roe_flux<in_n_fields,in_n_dims> (q_l,q_r,norm,fn,in_gamma);
    
    // Store transformed flux
    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr[thread_id]));     
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
	    (*(in_norm_tconf_fpts_l_ptr[thread_id+i*stride]))=jac*fn[i];

    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr[thread_id]));     
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
	    (*(in_norm_tconf_fpts_r_ptr[thread_id+i*stride]))=-jac*fn[i];

    // Viscous solution correction
    if(in_viscous)
    {
      if(in_vis_riemann_solve_type==0)
        ldg_solution<in_n_dims,in_n_fields,0> (q_l,q_r,norm,q_c,in_pen_fact);

      #pragma unroll
      for (int i=0;i<in_n_fields;i++) 
        (*(in_delta_disu_fpts_l_ptr[thread_id+i*stride])) = (q_c[i]-q_l[i]);   

      #pragma unroll
      for (int i=0;i<in_n_fields;i++) 
        (*(in_delta_disu_fpts_r_ptr[thread_id+i*stride])) = (q_c[i]-q_r[i]);
    }

  }
}


template <int in_n_dims, int in_vis_riemann_solve_type>
__global__ void calc_norm_tconinvf_fpts_lax_friedrich_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_norm_tconf_fpts_r_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr, double** in_norm_fpts_ptr, double** in_delta_disu_fpts_l_ptr, double** in_delta_disu_fpts_r_ptr, double in_pen_fact, int in_viscous, double wave_speed_x, double wave_speed_y, double wave_speed_z, double lambda)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int stride = in_n_fpts_per_inter*in_n_inters;
	
  double q_l; 
  double q_r; 
  double fn,u_av,u_diff;
  double norm_speed;
  double norm[in_n_dims];
  
  double q_c;
  double jac;

	if(thread_id<stride)
  {  
    // Compute left state solution
  	q_l=(*(in_disu_fpts_l_ptr[thread_id])); 

    // Compute right state solution
  	q_r=(*(in_disu_fpts_r_ptr[thread_id])); 
 
    // Compute normal
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  norm[i]=*(in_norm_fpts_ptr[thread_id + i*stride]);

    u_av = 0.5*(q_r+q_l);
    u_diff = q_l-q_r; 

    norm_speed=0.;
    if (in_n_dims==2)
      norm_speed += wave_speed_x*norm[0] + wave_speed_y*norm[1];
    else if (in_n_dims==3)
      norm_speed += wave_speed_x*norm[0] + wave_speed_y*norm[1] + wave_speed_z*norm[2];
	  		
    // Compute common interface flux
    fn = 0.;
    if (in_n_dims==2)
      fn += (wave_speed_x*norm[0] + wave_speed_y*norm[1])*u_av;
    else if (in_n_dims==3)
      fn += (wave_speed_x*norm[0] + wave_speed_y*norm[1] + wave_speed_z*norm[2])*u_av;
    fn += 0.5*lambda*abs(norm_speed)*u_diff;

    // Store transformed flux
    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr[thread_id]));     
	  (*(in_norm_tconf_fpts_l_ptr[thread_id]))=jac*fn;

    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr[thread_id]));     
	  (*(in_norm_tconf_fpts_r_ptr[thread_id]))=-jac*fn;

    // viscous solution correction
    if(in_viscous)
    {
      //if(in_vis_riemann_solve_type==0)
      //  ldg_solution<in_n_dims,1,0> (&q_l,&q_r,norm,&q_c,in_pen_fact);

			if(in_n_dims==2)
			{
    		if ((norm[0]+norm[1]) < 0.)
    	  	in_pen_fact=-in_pen_fact;
			}
			if(in_n_dims==3)
			{
    		if ((norm[0]+norm[1]+sqrt(2.)*norm[2]) < 0.)
    	  	in_pen_fact=-in_pen_fact;
			}

      q_c = 0.5*(q_l+q_r) - in_pen_fact*(q_l-q_r);

      //printf("%4.2f \n", q_c);

      (*(in_delta_disu_fpts_l_ptr[thread_id])) = (q_c-q_l);   
     
      (*(in_delta_disu_fpts_r_ptr[thread_id])) = (q_c-q_r);
    }
  }

}


// kernel to calculate normal transformed continuous inviscid flux at the flux points at boundaries
template<int in_n_dims, int in_n_fields, int in_riemann_solve_type, int in_vis_riemann_solve_type>
__global__ void calc_norm_tconinvf_fpts_boundary_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_norm_fpts_ptr, double** in_loc_fpts_ptr, int* in_boundary_type, double* in_bdy_params, double** in_delta_disu_fpts_l_ptr, double in_gamma, double in_R_ref, int in_viscous, double in_time_bound, double in_wave_speed_x, double in_wave_speed_y, double in_wave_speed_z, double in_lambda, int in_equation)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int stride = in_n_fpts_per_inter*in_n_inters;

  int bdy_spec;
 
  double q_l[in_n_fields]; 
  double q_r[in_n_fields]; 
  double fn[in_n_fields];
  double norm[in_n_dims];
  double loc[in_n_dims];
  double q_c[in_n_fields];

  double jac;

	if(thread_id<stride)
  {  
    // Compute left solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_l[i]=(*(in_disu_fpts_l_ptr[thread_id+i*stride])); 

    // Compute normal
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  norm[i]=*(in_norm_fpts_ptr[thread_id + i*stride]);

    // Compute location
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  loc[i]=*(in_loc_fpts_ptr[thread_id + i*stride]);

    // Set boundary condition
    bdy_spec = in_boundary_type[thread_id/in_n_fpts_per_inter];
    set_inv_boundary_conditions_kernel<in_n_dims,in_n_fields>(bdy_spec,q_l,q_r,norm,loc,in_bdy_params,in_gamma, in_R_ref, in_time_bound, in_equation);

    if (bdy_spec==16) // Dual consistent
    {
    //  right_flux<in_n_fields,in_n_dims> (q_r,norm,fn,in_gamma);
        roe_flux<in_n_fields,in_n_dims> (q_l,q_r,norm,fn,in_gamma);
    }
    else
    {
      if (in_riemann_solve_type==0)
        rusanov_flux<in_n_fields,in_n_dims> (q_l,q_r,norm,fn,in_gamma);
			else if (in_riemann_solve_type==1)
				lax_friedrichs_flux<in_n_dims> (q_l,q_r,norm,fn,in_wave_speed_x,in_wave_speed_y,in_wave_speed_z,in_lambda);
      else if (in_riemann_solve_type==2)
        roe_flux<in_n_fields,in_n_dims> (q_l,q_r,norm,fn,in_gamma);
    }

    // Store transformed flux
    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr[thread_id]));     
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
	    (*(in_norm_tconf_fpts_l_ptr[thread_id+i*stride]))=jac*fn[i];

    // Viscous solution correction
    if(in_viscous)
    {
      if(bdy_spec == 12 || bdy_spec == 14) // Adiabatic
      {
        if (in_vis_riemann_solve_type==0)
          ldg_solution<in_n_dims,in_n_fields,2> (q_l,q_r,norm,q_c,0);
      }
      else        
      {
        if (in_vis_riemann_solve_type==0)
          ldg_solution<in_n_dims,in_n_fields,1> (q_l,q_r,norm,q_c,0);
      }
      
      #pragma unroll
      for (int i=0;i<in_n_fields;i++) 
        (*(in_delta_disu_fpts_l_ptr[thread_id+i*stride])) = (q_c[i]-q_l[i]);
    }

  }
}


// gpu kernel to calculate transformed discontinuous viscous flux at solution points
template<int in_n_dims, int in_n_fields, int in_n_comp>
__global__ void calc_tdisvisf_upts_NS_gpu_kernel(int in_n_upts_per_ele, int in_n_eles, int in_ele_type, double in_filter_ratio, int LES, int SGS_model, double* Leonard_mom, double* Leonard_ene, double* in_disu_upts_ptr, double* out_tdisf_upts_ptr, double* in_grad_disu_upts_ptr, double* in_detjac_upts_ptr, double* in_inv_detjac_mul_jac_upts_ptr, double in_gamma, double in_prandtl, double in_rt_inf, double in_mu_inf, double in_c_sth, double in_fix_vis)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;

  double q[in_n_fields];
  double f[in_n_dims];
  double sgsf[in_n_dims];	// subgrid-scale flux set by SGS model
  double met[in_n_dims][in_n_dims];
  double stensor[in_n_comp]; // viscous stress tensor
  double straintensor[in_n_comp]; // for SGS models
  double sdtensor[in_n_comp]; // for WALE SGS model
	double lmtensor[in_n_comp]; // local Leonard tensor for momentum
	double letensor[in_n_dims]; // local Leonard tensor for energy
  double grad_ene[in_n_dims];
  double grad_vel[in_n_dims*in_n_dims];
  double grad_q[in_n_fields*in_n_dims];  
  double inte, mu, jac, delta;

	int i, j, ind;
	int stride = in_n_upts_per_ele*in_n_eles;

 	if(thread_id<(in_n_upts_per_ele*in_n_eles))
 	{
    // Physical solution
    #pragma unroll
    for (i=0;i<in_n_fields;i++) {
      q[i] = in_disu_upts_ptr[thread_id + i*stride];
		}

    #pragma unroll
    for (i=0;i<in_n_dims;i++) {
      #pragma unroll
      for (j=0;j<in_n_dims;j++) {
        met[j][i] = in_inv_detjac_mul_jac_upts_ptr[thread_id + (i*in_n_dims+j)*stride];
    	}
		}

    // Physical gradient
    #pragma unroll
    for (i=0;i<in_n_fields;i++)
    {
      ind = thread_id + i*stride;
      grad_q[i*in_n_dims + 0] = in_grad_disu_upts_ptr[ind];
      grad_q[i*in_n_dims + 1] = in_grad_disu_upts_ptr[ind + stride*in_n_fields];
      
      if(in_n_dims==3)
        grad_q[i*in_n_dims + 2] = in_grad_disu_upts_ptr[ind + 2*stride*in_n_fields];
    }

    // Flux prep
    vis_NS_flux<in_n_dims>(q, grad_q, grad_vel, grad_ene, stensor, f, &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, -1);

		int index;

    // Flux computation
    #pragma unroll
    for (i=0;i<in_n_fields;i++) {

      vis_NS_flux<in_n_dims>(q, grad_q, grad_vel, grad_ene, stensor, f, &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, i);

			// If LES, calculate SGS stress tensor
			if(LES) {
				// Calculate strain rate tensor from viscous stress tensor
				#pragma unroll
				for (j=0;j<in_n_comp;j++)
					straintensor[j] = stensor[j]/2.0/mu;

				// Calculate filter width
				jac = in_detjac_upts_ptr[thread_id];

				delta = SGS_filter_width(jac, in_ele_type, in_n_dims, in_filter_ratio);

				// Local Leonard tensors
				#pragma unroll
				for (j=0;j<in_n_dims;j++) {
					// energy
					letensor[j] = Leonard_ene[thread_id + j*stride];
				}

				#pragma unroll
				for (j=0;j<in_n_comp;j++) {
					// momentum
					lmtensor[j] = Leonard_mom[thread_id + j*stride];
				}

				SGS_flux_kernel<in_n_dims>(q, grad_q, grad_vel, grad_ene, sdtensor, straintensor, lmtensor, letensor, sgsf, SGS_model, delta, in_gamma, i);
				//printf("Leonard2 = %10.10f, %10.10f, %10.10f\n", lmtensor[0], lmtensor[1], lmtensor[2]);
				//printf("sim flux: %10.10f\n", sgsf);

				// Add SGS flux to viscous flux
				#pragma unroll
				for (j=0;j<in_n_dims;j++) {
					f[j] += sgsf[j];
				}
			}

      index = thread_id+i*stride;
      
      if(in_n_dims==2) {
		    out_tdisf_upts_ptr[index                   ] += met[0][0]*f[0] + met[0][1]*f[1]; 
		    out_tdisf_upts_ptr[index+stride*in_n_fields] += met[1][0]*f[0] + met[1][1]*f[1]; 
      }
      else if(in_n_dims==3) {
			  out_tdisf_upts_ptr[index                     ] += met[0][0]*f[0] + met[0][1]*f[1] + met[0][2]*f[2]; 
			  out_tdisf_upts_ptr[index+  stride*in_n_fields] += met[1][0]*f[0] + met[1][1]*f[1] + met[1][2]*f[2]; 
			  out_tdisf_upts_ptr[index+2*stride*in_n_fields] += met[2][0]*f[0] + met[2][1]*f[1] + met[2][2]*f[2]; 
      }
    }

  }
}


// gpu kernel to calculate transformed discontinuous viscous flux at solution points
template<int in_n_dims>
__global__ void calc_tdisvisf_upts_AD_gpu_kernel(int in_n_upts_per_ele, int in_n_eles, double* in_disu_upts_ptr, double* out_tdisf_upts_ptr, double* in_grad_disu_upts_ptr, double* in_detjac_upts_ptr, double* in_inv_detjac_mul_jac_upts_ptr, double diff_coeff)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;

  double f[in_n_dims];	
  double met[in_n_dims][in_n_dims];
  double grad_q[in_n_dims];

  int ind;
  int index;
	int stride = in_n_upts_per_ele*in_n_eles;

 	if(thread_id<(in_n_upts_per_ele*in_n_eles))
 	{
    // Metric terms
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
      #pragma unroll
      for (int j=0;j<in_n_dims;j++) 
        met[j][i] = in_inv_detjac_mul_jac_upts_ptr[thread_id + (i*in_n_dims+j)*stride];
    
    // Physical gradient
    ind = thread_id;
    grad_q[0] = in_grad_disu_upts_ptr[ind];
    grad_q[1] = in_grad_disu_upts_ptr[ind + stride];
      
    if(in_n_dims==3)
      grad_q[2] = in_grad_disu_upts_ptr[ind + 2*stride];


    // Flux computation
    f[0] = -diff_coeff*grad_q[0];
    f[1] = -diff_coeff*grad_q[1];
    
    if(in_n_dims==3)
      f[2] = -diff_coeff*grad_q[2];    

    index = thread_id;
      
    if(in_n_dims==2) {
      out_tdisf_upts_ptr[index       ] += met[0][0]*f[0] + met[0][1]*f[1]; 
      out_tdisf_upts_ptr[index+stride] += met[1][0]*f[0] + met[1][1]*f[1]; 
    }
    else if(in_n_dims==3) {
      out_tdisf_upts_ptr[index         ] += met[0][0]*f[0] + met[0][1]*f[1] + met[0][2]*f[2]; 
      out_tdisf_upts_ptr[index+  stride] += met[1][0]*f[0] + met[1][1]*f[1] + met[1][2]*f[2]; 
      out_tdisf_upts_ptr[index+2*stride] += met[2][0]*f[0] + met[2][1]*f[1] + met[2][2]*f[2]; 
    }

  }
}

// gpu kernel to calculate transformed discontinuous viscous flux at solution points
template<int in_n_dims, int in_n_fields>
__global__ void transform_grad_disu_upts_kernel(int in_n_upts_per_ele, int in_n_eles, double* in_grad_disu_upts_ptr, double* in_detjac_upts_ptr, double* in_inv_detjac_mul_jac_upts_ptr)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;

  double dq[in_n_dims];
  double met[in_n_dims][in_n_dims];

  double jac;
  int ind;

	int stride = in_n_upts_per_ele*in_n_eles;

 	if(thread_id<(in_n_upts_per_ele*in_n_eles))
 	{
    // Obtain metric terms
    jac = in_detjac_upts_ptr[thread_id];

    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
      #pragma unroll
      for (int j=0;j<in_n_dims;j++) 
        met[j][i] = in_inv_detjac_mul_jac_upts_ptr[thread_id + (i*in_n_dims+j)*stride];
    
    // Compute physical gradient
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
    {
      ind = thread_id + i*stride;
      dq[0] = in_grad_disu_upts_ptr[ind];
      dq[1] = in_grad_disu_upts_ptr[ind + stride*in_n_fields];
    
      if(in_n_dims==2)
      {
        in_grad_disu_upts_ptr[ind                   ] = (1./jac)*(dq[0]*met[0][0] + dq[1]*met[1][0]);
        in_grad_disu_upts_ptr[ind+stride*in_n_fields] = (1./jac)*(dq[0]*met[0][1] + dq[1]*met[1][1]);
      }
      if(in_n_dims==3)
      {
        dq[2] = in_grad_disu_upts_ptr[ind + 2*stride*in_n_fields];
        
        in_grad_disu_upts_ptr[ind                     ] = (1./jac)*(dq[0]*met[0][0] + dq[1]*met[1][0] + dq[2]*met[2][0]);
        in_grad_disu_upts_ptr[ind+stride*in_n_fields  ] = (1./jac)*(dq[0]*met[0][1] + dq[1]*met[1][1] + dq[2]*met[2][1]);
        in_grad_disu_upts_ptr[ind+2*stride*in_n_fields] = (1./jac)*(dq[0]*met[0][2] + dq[1]*met[1][2] + dq[2]*met[2][2]);
      }
    }
  }

}


// gpu kernel to calculate normal transformed continuous viscous flux at the flux points
template <int in_n_dims, int in_n_fields, int in_n_comp, int in_vis_riemann_solve_type>
__global__ void calc_norm_tconvisf_fpts_NS_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_grad_disu_fpts_l_ptr, double** in_grad_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_norm_tconf_fpts_r_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr, double** in_norm_fpts_ptr, double in_pen_fact, double in_tau, double in_gamma, double in_prandtl, double in_rt_inf, double in_mu_inf, double in_c_sth, double in_fix_vis)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int stride = in_n_fpts_per_inter*in_n_inters;
	
  double q_l[in_n_fields]; 
  double q_r[in_n_fields]; 
  double f_l[in_n_fields][in_n_dims]; 
  double f_r[in_n_fields][in_n_dims]; 
  double f_c[in_n_fields][in_n_dims]; 
  
  double fn[in_n_fields];
  double norm[in_n_dims];
  
  double grad_ene[in_n_dims];
  double grad_vel[in_n_dims*in_n_dims];
  double grad_q[in_n_fields*in_n_dims];
  
  double stensor[in_n_comp];
  
  double jac;
  double inte, mu;

	if(thread_id<stride)
  {  
    // Left solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_l[i]=(*(in_disu_fpts_l_ptr[thread_id+i*stride])); 

    // Left solution gradient
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
    {
      #pragma unroll
      for(int j=0;j<in_n_dims;j++)
        grad_q[i*in_n_dims + j] = *(in_grad_disu_fpts_l_ptr[thread_id + (j*in_n_fields + i)*stride]);
    }

    // Normal vector
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  norm[i]=*(in_norm_fpts_ptr[thread_id + i*stride]);
    
    // Left flux prep
    vis_NS_flux<in_n_dims>(q_l, grad_q, grad_vel, grad_ene, stensor, NULL, &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, -1);

    // Left flux computation
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
      vis_NS_flux<in_n_dims>(q_l, grad_q, grad_vel, grad_ene, stensor, f_l[i], &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, i);
    

    // Right solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_r[i]=(*(in_disu_fpts_r_ptr[thread_id+i*stride])); 

    // Right solution gradient
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
    {
      #pragma unroll
      for(int j=0;j<in_n_dims;j++)
        grad_q[i*in_n_dims + j] = *(in_grad_disu_fpts_r_ptr[thread_id + (j*in_n_fields + i)*stride]);
    }
    
    // Right flux prep
    vis_NS_flux<in_n_dims>(q_r, grad_q, grad_vel, grad_ene, stensor, NULL, &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, -1);
    
    // Right flux computation
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
      vis_NS_flux<in_n_dims>(q_r, grad_q, grad_vel, grad_ene, stensor, f_r[i], &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, i);

    // Compute common flux
    if(in_vis_riemann_solve_type == 0)
    {
      #pragma unroll
      for (int i=0;i<in_n_fields;i++)
        ldg_flux<in_n_dims,0>(q_l[i],q_r[i],f_l[i],f_r[i],f_c[i],norm,in_pen_fact,in_tau);
    }

    // Compute common normal flux
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
    {
      fn[i] = f_c[i][0]*norm[0];
      #pragma unroll
      for (int j=1;j<in_n_dims;j++)
        fn[i] += f_c[i][j]*norm[j];
    }

    // Store transformed flux
    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr[thread_id]));     
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
	    (*(in_norm_tconf_fpts_l_ptr[thread_id+i*stride]))+=jac*fn[i];

    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr[thread_id]));     
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
	    (*(in_norm_tconf_fpts_r_ptr[thread_id+i*stride]))+=-jac*fn[i];

  }
}


// gpu kernel to calculate normal transformed continuous viscous flux at the flux points
template <int in_n_dims>
__global__ void calc_norm_tconvisf_fpts_AD_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_grad_disu_fpts_l_ptr, double** in_grad_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_norm_tconf_fpts_r_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr, double** in_norm_fpts_ptr, double in_pen_fact, double in_tau, double diff_coeff)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int stride = in_n_fpts_per_inter*in_n_inters;
	
  double q_l; 
  double q_r; 
  double f_l[in_n_dims]; 
  double f_r[in_n_dims]; 
  double f_c[in_n_dims]; 
  
  double fn;
  double norm[in_n_dims];
  
  double grad_q[in_n_dims];
  double jac;

	if(thread_id<stride)
  {  
    // Left solution
    q_l=(*(in_disu_fpts_l_ptr[thread_id])); 

    // Left solution gradient
    #pragma unroll
    for(int j=0;j<in_n_dims;j++)
      grad_q[j] = *(in_grad_disu_fpts_l_ptr[thread_id + j*stride]);

    // Normal vector
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  norm[i]=*(in_norm_fpts_ptr[thread_id + i*stride]);

    // Left flux computation
    f_l[0] = -diff_coeff*grad_q[0];
    f_l[1] = -diff_coeff*grad_q[1];
    
    if (in_n_dims==3)
      f_l[2] = -diff_coeff*grad_q[2];


    // Right solution
    q_r=(*(in_disu_fpts_r_ptr[thread_id])); 

    // Right solution gradient
    #pragma unroll
    for(int j=0;j<in_n_dims;j++)
      grad_q[j] = *(in_grad_disu_fpts_r_ptr[thread_id + j*stride]);
    
    // Right flux computation
    f_r[0] = -diff_coeff*grad_q[0];
    f_r[1] = -diff_coeff*grad_q[1];
    
    if (in_n_dims==3)
      f_r[2] = -diff_coeff*grad_q[2];

    // Compute common flux
    ldg_flux<in_n_dims,0>(q_l,q_r,f_l,f_r,f_c,norm,in_pen_fact,in_tau);

    // Compute common normal flux
    fn = f_c[0]*norm[0];
    #pragma unroll
    for (int j=1;j<in_n_dims;j++)
      fn += f_c[j]*norm[j];

    // Store transformed flux
    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr[thread_id]));     
    (*(in_norm_tconf_fpts_l_ptr[thread_id]))+=jac*fn;

    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr[thread_id]));     
    (*(in_norm_tconf_fpts_r_ptr[thread_id]))+=-jac*fn;

  }
}



// kernel to calculate normal transformed continuous viscous flux at the flux points at boundaries
template<int in_n_dims, int in_n_fields, int in_n_comp, int in_vis_riemann_solve_type>
__global__ void calc_norm_tconvisf_fpts_boundary_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_grad_disu_fpts_l_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_norm_fpts_ptr, double** in_loc_fpts_ptr, int* in_boundary_type, double* in_bdy_params, double** in_delta_disu_fpts_l_ptr, double in_R_ref, double in_pen_fact, double in_tau, double in_gamma, double in_prandtl, double in_rt_inf, double in_mu_inf, double in_c_sth, double in_fix_vis, double in_time_bound, int in_equation, double diff_coeff)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int stride = in_n_fpts_per_inter*in_n_inters;

  int bdy_spec;

  double q_l[in_n_fields]; 
  double q_r[in_n_fields]; 
  
  double f[in_n_fields][in_n_dims]; 
  double f_c[in_n_fields][in_n_dims]; 
  
  double fn[in_n_fields];
  double norm[in_n_dims];
  double loc[in_n_dims];
  
  double grad_ene[in_n_dims];
  double grad_vel[in_n_dims*in_n_dims];
  double grad_q[in_n_fields*in_n_dims];
  
  double stensor[in_n_comp];

  double jac;
  double inte, mu;

	if(thread_id<stride)
  {  
    // Left solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_l[i]=(*(in_disu_fpts_l_ptr[thread_id+i*stride])); 
      
    // Left solution gradient
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
    {
      #pragma unroll
      for(int j=0;j<in_n_dims;j++)
        grad_q[i*in_n_dims + j] = *(in_grad_disu_fpts_l_ptr[thread_id + (j*in_n_fields + i)*stride]);
    }
    
    // Normal vector
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  norm[i]=*(in_norm_fpts_ptr[thread_id + i*stride]);
    
    // Compute location
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  loc[i]=*(in_loc_fpts_ptr[thread_id + i*stride]);
    
    // Right solution
    bdy_spec = in_boundary_type[thread_id/in_n_fpts_per_inter];
    set_inv_boundary_conditions_kernel<in_n_dims,in_n_fields>(bdy_spec,q_l,q_r,norm,loc,in_bdy_params,in_gamma,in_R_ref,in_time_bound,in_equation);


    // Compute common flux
    if(bdy_spec == 12 || bdy_spec == 14) 
    {
      // Right solution gradient
      set_vis_boundary_conditions_kernel<in_n_dims,in_n_fields>(bdy_spec,q_l,q_r,grad_q,norm,loc,in_bdy_params,in_gamma,in_R_ref,in_time_bound,in_equation);
    
			if(in_equation==0)
			{
      	// Right flux prep
      	vis_NS_flux<in_n_dims>(q_r, grad_q, grad_vel, grad_ene, stensor, NULL, &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, -1);
    
      	// Right flux computation
      	#pragma unroll
      	for (int i=0;i<in_n_fields;i++)
        	vis_NS_flux<in_n_dims>(q_r, grad_q, grad_vel, grad_ene, stensor, f[i], &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, i);
			}
			if(in_equation==1)
			{
    		f[0][0] = -diff_coeff*grad_q[0];
    		f[0][1] = -diff_coeff*grad_q[1];
    
				if(in_n_dims==3)
      		f[0][2] = -diff_coeff*grad_q[2];
			}
	
      if (in_vis_riemann_solve_type==0)
      {
        #pragma unroll
        for (int i=0;i<in_n_fields;i++)
          ldg_flux<in_n_dims,2>(q_l[i],q_r[i],NULL,f[i],f_c[i],norm,in_pen_fact,in_tau); // von Neumann
      }
    }
    else
    {
			if(in_equation==0)
			{
      	// Left flux prep
      	vis_NS_flux<in_n_dims>(q_l, grad_q, grad_vel, grad_ene, stensor, NULL, &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, -1);

      	// Left flux computation
      	#pragma unroll
      	for (int i=0;i<in_n_fields;i++)
        	vis_NS_flux<in_n_dims>(q_l, grad_q, grad_vel, grad_ene, stensor, f[i], &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, i);
			}
			if(in_equation==1)
			{
    		f[0][0] = -diff_coeff*grad_q[0];
    		f[0][1] = -diff_coeff*grad_q[1];
    
				if(in_n_dims==3)
      		f[0][2] = -diff_coeff*grad_q[2];
			}        

      if (in_vis_riemann_solve_type==0)
      {
        #pragma unroll
        for (int i=0;i<in_n_fields;i++)
          ldg_flux<in_n_dims,1>(q_l[i],q_r[i],f[i],NULL,f_c[i],norm,in_pen_fact,in_tau); // Dirichlet
      }
    }

    // compute common normal flux
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
    {
      fn[i] = f_c[i][0]*norm[0];
      #pragma unroll
      for (int j=1;j<in_n_dims;j++)
        fn[i] += f_c[i][j]*norm[j];
    }

    // store transformed flux
    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr[thread_id]));     
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
	    (*(in_norm_tconf_fpts_l_ptr[thread_id+i*stride]))+=jac*fn[i];

  }
}


#ifdef _MPI

// gpu kernel to calculate normal transformed continuous inviscid flux at the flux points for mpi faces
template <int in_n_dims, int in_n_fields, int in_riemann_solve_type, int in_vis_riemann_solve_type>
__global__ void calc_norm_tconinvf_fpts_NS_mpi_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_norm_fpts_ptr, double** in_delta_disu_fpts_l_ptr, double in_gamma, double in_pen_fact, int in_viscous)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int stride = in_n_fpts_per_inter*in_n_inters;
	
  double q_l[in_n_fields]; 
  double q_r[in_n_fields]; 
  double fn[in_n_fields];
  double norm[in_n_dims];

  double q_c[in_n_fields];

  double jac;

	if(thread_id<stride)
  {  
    // Compute left state solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_l[i]=(*(in_disu_fpts_l_ptr[thread_id+i*stride])); 

    // Compute right state solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_r[i]=*(in_disu_fpts_r_ptr[thread_id+i*stride]);

    // Compute normal
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  norm[i]=*(in_norm_fpts_ptr[thread_id + i*stride]);

    if (in_riemann_solve_type==0)
      rusanov_flux<in_n_fields,in_n_dims> (q_l,q_r,norm,fn,in_gamma);
    else if (in_riemann_solve_type==2)
      roe_flux<in_n_fields,in_n_dims> (q_l,q_r,norm,fn,in_gamma);

    // Store transformed flux
    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr[thread_id]));     
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
	    (*(in_norm_tconf_fpts_l_ptr[thread_id+i*stride]))=jac*fn[i];

    // viscous solution correction
    if(in_viscous)
    {
      if(in_vis_riemann_solve_type==0)
        ldg_solution<in_n_dims,in_n_fields,0> (q_l,q_r,norm,q_c,in_pen_fact);

      #pragma unroll
      for (int i=0;i<in_n_fields;i++) 
      {
        (*(in_delta_disu_fpts_l_ptr[thread_id+i*stride])) = (q_c[i]-q_l[i]);   
      }
    }

  }
}


// gpu kernel to calculate normal transformed continuous viscous flux at the flux points
template <int in_n_dims, int in_n_fields, int in_n_comp, int in_vis_riemann_solve_type>
__global__ void calc_norm_tconvisf_fpts_NS_mpi_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_grad_disu_fpts_l_ptr, double** in_grad_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_norm_fpts_ptr, double in_pen_fact, double in_tau, double in_gamma, double in_prandtl, double in_rt_inf, double in_mu_inf, double in_c_sth, double in_fix_vis)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int stride = in_n_fpts_per_inter*in_n_inters;
	
  double q_l[in_n_fields]; 
  double q_r[in_n_fields]; 
  double f_l[in_n_fields][in_n_dims]; 
  double f_r[in_n_fields][in_n_dims]; 
  double f_c[in_n_fields][in_n_dims]; 
  
  double fn[in_n_fields];
  double norm[in_n_dims];
  
  double grad_ene[in_n_dims];
  double grad_vel[in_n_dims*in_n_dims];
  double grad_q[in_n_fields*in_n_dims];
  
  double stensor[in_n_comp];
  
  double jac;
  double inte, mu;

	if(thread_id<stride)
  {  
    // Left solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_l[i]=(*(in_disu_fpts_l_ptr[thread_id+i*stride])); 

    // Left solution gradient
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
    {
      #pragma unroll
      for(int j=0;j<in_n_dims;j++)
        grad_q[i*in_n_dims + j] = *(in_grad_disu_fpts_l_ptr[thread_id + (j*in_n_fields + i)*stride]);
    }

    // Normal vector
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  norm[i]=*(in_norm_fpts_ptr[thread_id + i*stride]);
    
    // Left flux prep
    vis_NS_flux<in_n_dims>(q_l, grad_q, grad_vel, grad_ene, stensor, NULL, &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, -1);

    // Left flux computation
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
      vis_NS_flux<in_n_dims>(q_l, grad_q, grad_vel, grad_ene, stensor, f_l[i], &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, i);


    // Right solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_r[i]=(*(in_disu_fpts_r_ptr[thread_id+i*stride]));// don't divide by jac, since points to buffer

    // Right solution gradient
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
    {
      #pragma unroll
      for(int j=0;j<in_n_dims;j++)
        grad_q[i*in_n_dims + j] = *(in_grad_disu_fpts_r_ptr[thread_id + (j*in_n_fields + i)*stride]);
    }
    
    // Right flux prep
    vis_NS_flux<in_n_dims>(q_r, grad_q, grad_vel, grad_ene, stensor, NULL, &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, -1);
    
    // Right flux computation
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
      vis_NS_flux<in_n_dims>(q_r, grad_q, grad_vel, grad_ene, stensor, f_r[i], &inte, &mu, in_prandtl, in_gamma, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis, i);

   
    // Compute common flux
    if(in_vis_riemann_solve_type == 0)
    {
      #pragma unroll
      for (int i=0;i<in_n_fields;i++)
        ldg_flux<in_n_dims,0>(q_l[i],q_r[i],f_l[i],f_r[i],f_c[i],norm,in_pen_fact,in_tau);
    }

    // Compute common normal flux
    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
    {
      fn[i] = f_c[i][0]*norm[0];
      #pragma unroll
      for (int j=1;j<in_n_dims;j++)
        fn[i] += f_c[i][j]*norm[j];
    }

    // Store transformed flux
    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr[thread_id]));     
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
	    (*(in_norm_tconf_fpts_l_ptr[thread_id+i*stride]))+=jac*fn[i];

  }
}


// gpu kernel to calculate normal transformed continuous viscous flux at the flux points
template <int in_n_dims>
__global__ void calc_norm_tconvisf_fpts_AD_mpi_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_grad_disu_fpts_l_ptr, double** in_grad_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_norm_fpts_ptr, double in_pen_fact, double in_tau, double diff_coeff)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int stride = in_n_fpts_per_inter*in_n_inters;
	
  double q_l; 
  double q_r; 
  double f_l[in_n_dims]; 
  double f_r[in_n_dims]; 
  double f_c[in_n_dims]; 
  
  double fn;
  double norm[in_n_dims];
  
  double grad_q[in_n_dims];
  double jac;

	if(thread_id<stride)
  {  
    // Left solution
    q_l=(*(in_disu_fpts_l_ptr[thread_id])); 

    // Left solution gradient
    #pragma unroll
    for(int j=0;j<in_n_dims;j++)
      grad_q[j] = *(in_grad_disu_fpts_l_ptr[thread_id + j*stride]);

    // Normal vector
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  norm[i]=*(in_norm_fpts_ptr[thread_id + i*stride]);

    // Left flux computation
    f_l[0] = -diff_coeff*grad_q[0];
    f_l[1] = -diff_coeff*grad_q[1];
    
    if (in_n_dims==3)
      f_l[2] = -diff_coeff*grad_q[2];


    // Right solution
    q_r=(*(in_disu_fpts_r_ptr[thread_id])); 

    // Right solution gradient
    #pragma unroll
    for(int j=0;j<in_n_dims;j++)
      grad_q[j] = *(in_grad_disu_fpts_r_ptr[thread_id + j*stride]);
    
    // Right flux computation
    f_r[0] = -diff_coeff*grad_q[0];
    f_r[1] = -diff_coeff*grad_q[1];
    
    if (in_n_dims==3)
      f_r[2] = -diff_coeff*grad_q[2];

    // Compute common flux
    ldg_flux<in_n_dims,0>(q_l,q_r,f_l,f_r,f_c,norm,in_pen_fact,in_tau);

    // Compute common normal flux
    fn = f_c[0]*norm[0];
    #pragma unroll
    for (int j=1;j<in_n_dims;j++)
      fn += f_c[j]*norm[j];

    // Store transformed flux
    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr[thread_id]));     
    (*(in_norm_tconf_fpts_l_ptr[thread_id]))+=jac*fn;

  }
}


template <int in_n_dims, int in_vis_riemann_solve_type>
__global__ void calc_norm_tconinvf_fpts_lax_friedrich_mpi_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_norm_fpts_ptr, double** in_delta_disu_fpts_l_ptr, double in_pen_fact, int in_viscous, double wave_speed_x, double wave_speed_y, double wave_speed_z, double lambda)
{
	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int stride = in_n_fpts_per_inter*in_n_inters;
	
  double q_l; 
  double q_r; 
  double fn,u_av,u_diff;
  double norm_speed;
  double norm[in_n_dims];
  
  double q_c;
  double jac;

	if(thread_id<stride)
  {  

   // Compute left state solution
  	q_l=(*(in_disu_fpts_l_ptr[thread_id])); 

    // Compute right state solution
  	q_r=(*(in_disu_fpts_r_ptr[thread_id]));

    // Compute normal
    #pragma unroll
    for (int i=0;i<in_n_dims;i++) 
  	  norm[i]=*(in_norm_fpts_ptr[thread_id + i*stride]);

    u_av = 0.5*(q_r+q_l);
    u_diff = q_l-q_r; 

    norm_speed=0.;
    if (in_n_dims==2)
      norm_speed += wave_speed_x*norm[0] + wave_speed_y*norm[1];
    else if (in_n_dims==3)
      norm_speed += wave_speed_x*norm[0] + wave_speed_y*norm[1] + wave_speed_z*norm[2];
	  		
    // Compute common interface flux
    fn = 0.;
    if (in_n_dims==2)
      fn += (wave_speed_x*norm[0] + wave_speed_y*norm[1])*u_av;
    else if (in_n_dims==3)
      fn += (wave_speed_x*norm[0] + wave_speed_y*norm[1] + wave_speed_z*norm[2])*u_av;
    fn += 0.5*lambda*abs(norm_speed)*u_diff;

    // Store transformed flux
    jac = (*(in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr[thread_id]));     
	  (*(in_norm_tconf_fpts_l_ptr[thread_id]))=jac*fn;

    // viscous solution correction
    if(in_viscous)
    {
			if(in_n_dims==2)
			{
    		if ((norm[0]+norm[1]) < 0.)
    	  	in_pen_fact=-in_pen_fact;
			}
			if(in_n_dims==3)
			{
    		if ((norm[0]+norm[1]+sqrt(2.)*norm[2]) < 0.)
    	  	in_pen_fact=-in_pen_fact;
			}

      q_c = 0.5*(q_l+q_r) - in_pen_fact*(q_l-q_r);

      /*
      if(in_vis_riemann_solve_type==0)
        ldg_solution<in_n_dims,1,0> (&q_l,&q_r,norm,&q_c,in_pen_fact);
      */

      (*(in_delta_disu_fpts_l_ptr[thread_id])) = (q_c-q_l);   
    }
  }
}


template <int in_n_fields>
__global__ void  pack_out_buffer_disu_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_disu_fpts_l_ptr, double* in_out_buffer_disu_ptr)
{

  double q_l[in_n_fields];

	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int inter = thread_id/(in_n_fpts_per_inter);
  const int fpt = thread_id - inter*in_n_fpts_per_inter;
  const int stride=in_n_fpts_per_inter*in_n_inters;

  if (thread_id < stride)
  {
    // Compute left state solution
    #pragma unroll
    for (int i=0;i<in_n_fields;i++) 
  	  q_l[i]=(*(in_disu_fpts_l_ptr[thread_id+i*stride])); 

    #pragma unroll
    for (int i=0;i<in_n_fields;i++)
      in_out_buffer_disu_ptr[inter*in_n_fpts_per_inter*in_n_fields+i*in_n_fpts_per_inter+fpt]=q_l[i];

  }

}


template <int in_n_fields, int in_n_dims>
__global__ void  pack_out_buffer_grad_disu_gpu_kernel(int in_n_fpts_per_inter, int in_n_inters, double** in_grad_disu_fpts_l_ptr, double* in_out_buffer_grad_disu_ptr)
{

  double dq[in_n_fields][in_n_dims];

	const int thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  const int inter = thread_id/(in_n_fpts_per_inter);
  const int fpt = thread_id - inter*in_n_fpts_per_inter;
  const int stride=in_n_fpts_per_inter*in_n_inters;

  if (thread_id < stride)
  {
    // Compute left state solution
    #pragma unroll
    for (int j=0;j<in_n_dims;j++)
      #pragma unroll
      for (int i=0;i<in_n_fields;i++) 
  	    dq[i][j]=(*(in_grad_disu_fpts_l_ptr[thread_id+(j*in_n_fields+i)*stride]));

    #pragma unroll
    for (int j=0;j<in_n_dims;j++)
      #pragma unroll
      for (int i=0;i<in_n_fields;i++)
        in_out_buffer_grad_disu_ptr[inter*in_n_fpts_per_inter*in_n_fields*in_n_dims+j*in_n_fpts_per_inter*in_n_fields+i*in_n_fpts_per_inter+fpt]=dq[i][j];

  }

}

#endif



void RK45_update_kernel_wrapper(int in_n_upts_per_ele,int in_n_dims,int in_n_fields,int in_n_eles,double* in_disu0_upts_ptr,double* in_disu1_upts_ptr,double* in_div_tconf_upts_ptr, double* in_detjac_upts_ptr, double in_rk4a, double in_rk4b, double in_dt, double in_const_src_term)
{

	// HACK: fix 256 threads per block
	int n_blocks=((in_n_eles*in_n_upts_per_ele-1)/256)+1;

  if (in_n_fields==1)
  {
	  RK45_update_kernel <1> <<< n_blocks,256>>> (in_disu0_upts_ptr, in_div_tconf_upts_ptr, in_disu1_upts_ptr, in_detjac_upts_ptr, in_n_eles, in_n_upts_per_ele, in_rk4a, in_rk4b, in_dt, in_const_src_term);
  }
  else if (in_n_fields==4)
  {
	  RK45_update_kernel <4> <<< n_blocks,256>>> (in_disu0_upts_ptr, in_div_tconf_upts_ptr, in_disu1_upts_ptr, in_detjac_upts_ptr, in_n_eles, in_n_upts_per_ele, in_rk4a, in_rk4b, in_dt, in_const_src_term);
  }
  else if (in_n_fields==5)
  {
	  RK45_update_kernel <5> <<< n_blocks,256>>> (in_disu0_upts_ptr, in_div_tconf_upts_ptr, in_disu1_upts_ptr, in_detjac_upts_ptr, in_n_eles, in_n_upts_per_ele, in_rk4a, in_rk4b, in_dt, in_const_src_term);
  }
  else 
    FatalError("n_fields not supported");

}

void RK11_update_kernel_wrapper(int in_n_upts_per_ele,int in_n_dims,int in_n_fields,int in_n_eles,double* in_disu0_upts_ptr,double* in_div_tconf_upts_ptr, double* in_detjac_upts_ptr, double in_dt, double in_const_src_term)
{

	// HACK: fix 256 threads per block
	int n_blocks=((in_n_eles*in_n_upts_per_ele-1)/256)+1;

  if (in_n_fields==1)
  {
	  RK11_update_kernel <1> <<< n_blocks,256>>> (in_disu0_upts_ptr, in_div_tconf_upts_ptr, in_detjac_upts_ptr, in_n_eles, in_n_upts_per_ele, in_dt, in_const_src_term);
  }
  else if (in_n_fields==4)
  {
	  RK11_update_kernel <4> <<< n_blocks,256>>> (in_disu0_upts_ptr, in_div_tconf_upts_ptr, in_detjac_upts_ptr, in_n_eles, in_n_upts_per_ele, in_dt, in_const_src_term);
  }
  else if (in_n_fields==5)
  {
	  RK11_update_kernel <5> <<< n_blocks,256>>> (in_disu0_upts_ptr, in_div_tconf_upts_ptr, in_detjac_upts_ptr, in_n_eles, in_n_upts_per_ele, in_dt, in_const_src_term);
  }
  else 
    FatalError("n_fields not supported");

}


// wrapper for gpu kernel to calculate transformed discontinuous inviscid flux at solution points
void calc_tdisinvf_upts_gpu_kernel_wrapper(int in_n_upts_per_ele, int in_n_dims, int in_n_fields, int in_n_eles, double* in_disu_upts_ptr, double* out_tdisf_upts_ptr, double* in_detjac_upts_ptr, double* in_inv_detjac_mul_jac_upts_ptr, double in_gamma, int equation, double wave_speed_x, double wave_speed_y, double wave_speed_z)
{
	// HACK: fix 256 threads per block
	int n_blocks=((in_n_eles*in_n_upts_per_ele-1)/256)+1;

  check_cuda_error("Before", __FILE__, __LINE__);

  if (equation==0)
  {
    if (in_n_dims==2)
	    calc_tdisinvf_upts_NS_gpu_kernel<2,4> <<<n_blocks,256>>>(in_n_upts_per_ele,in_n_eles,in_disu_upts_ptr,out_tdisf_upts_ptr,in_detjac_upts_ptr,in_inv_detjac_mul_jac_upts_ptr,in_gamma);
    else if (in_n_dims==3)
	    calc_tdisinvf_upts_NS_gpu_kernel<3,5> <<<n_blocks,256>>>(in_n_upts_per_ele,in_n_eles,in_disu_upts_ptr,out_tdisf_upts_ptr,in_detjac_upts_ptr,in_inv_detjac_mul_jac_upts_ptr,in_gamma);
    else
		  FatalError("ERROR: Invalid number of dimensions ... ");
  }
  else if (equation==1)
  {
    if (in_n_dims==2)
	    calc_tdisinvf_upts_AD_gpu_kernel<2> <<<n_blocks,256>>>(in_n_upts_per_ele,in_n_eles,in_disu_upts_ptr,out_tdisf_upts_ptr,in_detjac_upts_ptr,in_inv_detjac_mul_jac_upts_ptr,wave_speed_x,wave_speed_y,wave_speed_z);
    else if (in_n_dims==3)
	    calc_tdisinvf_upts_AD_gpu_kernel<3> <<<n_blocks,256>>>(in_n_upts_per_ele,in_n_eles,in_disu_upts_ptr,out_tdisf_upts_ptr,in_detjac_upts_ptr,in_inv_detjac_mul_jac_upts_ptr,wave_speed_x,wave_speed_y,wave_speed_z);
    else
		  FatalError("ERROR: Invalid number of dimensions ... ");
  }
  else 
  {
    FatalError("equation not recognized");
  }

  check_cuda_error("After",__FILE__, __LINE__);
}



// wrapper for gpu kernel to calculate normal transformed continuous inviscid flux at the flux points
void calc_norm_tconinvf_fpts_gpu_kernel_wrapper(int in_n_fpts_per_inter, int in_n_dims, int in_n_fields, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_norm_tconinvf_fpts_l_ptr, double** in_norm_tconinvf_fpts_r_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr, double** in_norm_fpts_ptr, int in_riemann_solve_type, double** in_delta_disu_fpts_l_ptr, double** in_delta_disu_fpts_r_ptr, double in_gamma, double in_pen_fact, int in_viscous, int in_vis_riemann_solve_type, double wave_speed_x, double wave_speed_y, double wave_speed_z, double lambda)
{
	// HACK: fix 256 threads per block
	int n_blocks=((in_n_inters*in_n_fpts_per_inter-1)/256)+1;

  check_cuda_error("Before", __FILE__, __LINE__);

  if (in_riemann_solve_type==0) // Rusanov 
  {
    if(in_vis_riemann_solve_type==0) //LDG
    {
      if (in_n_dims==2)
	      calc_norm_tconinvf_fpts_NS_gpu_kernel<2,4,0,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconinvf_fpts_l_ptr,in_norm_tconinvf_fpts_r_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_delta_disu_fpts_r_ptr,in_gamma,in_pen_fact,in_viscous);
      else if (in_n_dims==3)
	      calc_norm_tconinvf_fpts_NS_gpu_kernel<3,5,0,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconinvf_fpts_l_ptr,in_norm_tconinvf_fpts_r_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_delta_disu_fpts_r_ptr,in_gamma,in_pen_fact,in_viscous);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized ... ");
  }
  else if ( in_riemann_solve_type==2) // Roe
  {
    if(in_vis_riemann_solve_type==0) //LDG
    {  
      if (in_n_dims==2)
	      calc_norm_tconinvf_fpts_NS_gpu_kernel<2,4,2,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconinvf_fpts_l_ptr,in_norm_tconinvf_fpts_r_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_delta_disu_fpts_r_ptr,in_gamma,in_pen_fact,in_viscous);
      else if (in_n_dims==3)
	      calc_norm_tconinvf_fpts_NS_gpu_kernel<3,5,2,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconinvf_fpts_l_ptr,in_norm_tconinvf_fpts_r_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_delta_disu_fpts_r_ptr,in_gamma,in_pen_fact,in_viscous);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized ... ");
  }
  else if (in_riemann_solve_type==1) // Lax-Friedrich
  {
    if(in_vis_riemann_solve_type==0) //LDG
    {
      if (in_n_dims==2)
	      calc_norm_tconinvf_fpts_lax_friedrich_gpu_kernel<2,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconinvf_fpts_l_ptr,in_norm_tconinvf_fpts_r_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_delta_disu_fpts_r_ptr,in_pen_fact,in_viscous,wave_speed_x,wave_speed_y,wave_speed_z,lambda);
      else if (in_n_dims==3)
	      calc_norm_tconinvf_fpts_lax_friedrich_gpu_kernel<3,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconinvf_fpts_l_ptr,in_norm_tconinvf_fpts_r_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_delta_disu_fpts_r_ptr,in_pen_fact,in_viscous,wave_speed_x,wave_speed_y,wave_speed_z,lambda);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized ... ");
  }
  else
    FatalError("ERROR: Riemann solver type not recognized ... ");
  
  check_cuda_error("After", __FILE__, __LINE__);
}

// wrapper for gpu kernel to calculate normal transformed continuous inviscid flux at the flux points at boundaries
void calc_norm_tconinvf_fpts_boundary_gpu_kernel_wrapper(int in_n_fpts_per_inter, int in_n_dims, int in_n_fields, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_norm_fpts_ptr, double** in_loc_fpts_ptr, int* in_boundary_type, double* in_bdy_params, int in_riemann_solve_type, double** in_delta_disu_fpts_l_ptr, double in_gamma, double in_R_ref, int in_viscous, int in_vis_riemann_solve_type, double in_time_bound, double in_wave_speed_x, double in_wave_speed_y, double in_wave_speed_z, double in_lambda, int in_equation)
{

  check_cuda_error("Before", __FILE__, __LINE__);
	// HACK: fix 256 threads per block
	int n_blocks=((in_n_inters*in_n_fpts_per_inter-1)/256)+1;

  if (in_riemann_solve_type==0)  // Rusanov
  {
    if (in_vis_riemann_solve_type==0) // LDG
    {
      if (in_n_dims==2)
	      calc_norm_tconinvf_fpts_boundary_gpu_kernel<2,4,0,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_loc_fpts_ptr,in_boundary_type, in_bdy_params, in_delta_disu_fpts_l_ptr, in_gamma, in_R_ref, in_viscous, in_time_bound, in_wave_speed_x, in_wave_speed_y, in_wave_speed_z, in_lambda, in_equation);
      else if (in_n_dims==3)
	      calc_norm_tconinvf_fpts_boundary_gpu_kernel<3,5,0,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_loc_fpts_ptr,in_boundary_type, in_bdy_params, in_delta_disu_fpts_l_ptr, in_gamma, in_R_ref, in_viscous, in_time_bound, in_wave_speed_x, in_wave_speed_y, in_wave_speed_z, in_lambda, in_equation);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized in bdy riemann solver");
  }
  else if (in_riemann_solve_type==1)  // Lax-Friedrichs
  {
    if (in_vis_riemann_solve_type==0) // LDG
    {
      if (in_n_dims==2)
	      calc_norm_tconinvf_fpts_boundary_gpu_kernel<2,1,1,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_loc_fpts_ptr,in_boundary_type, in_bdy_params, in_delta_disu_fpts_l_ptr, in_gamma, in_R_ref, in_viscous, in_time_bound, in_wave_speed_x, in_wave_speed_y, in_wave_speed_z, in_lambda, in_equation);
      else if (in_n_dims==3)
	      calc_norm_tconinvf_fpts_boundary_gpu_kernel<3,1,1,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_loc_fpts_ptr,in_boundary_type, in_bdy_params, in_delta_disu_fpts_l_ptr, in_gamma, in_R_ref, in_viscous, in_time_bound, in_wave_speed_x, in_wave_speed_y, in_wave_speed_z, in_lambda, in_equation);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized in bdy riemann solver");
  }
  else if (in_riemann_solve_type==2) // Roe
  {
    if (in_vis_riemann_solve_type==0) // LDG
    {
      if (in_n_dims==2)
	      calc_norm_tconinvf_fpts_boundary_gpu_kernel<2,4,2,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_loc_fpts_ptr,in_boundary_type, in_bdy_params, in_delta_disu_fpts_l_ptr, in_gamma, in_R_ref, in_viscous, in_time_bound, in_wave_speed_x, in_wave_speed_y, in_wave_speed_z, in_lambda, in_equation);
      else if (in_n_dims==3)
	      calc_norm_tconinvf_fpts_boundary_gpu_kernel<3,5,2,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_loc_fpts_ptr,in_boundary_type, in_bdy_params, in_delta_disu_fpts_l_ptr, in_gamma, in_R_ref, in_viscous, in_time_bound, in_wave_speed_x, in_wave_speed_y, in_wave_speed_z, in_lambda, in_equation);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized in bdy riemann solver");
  }
  else
  {
		FatalError("ERROR: Riemann solver type not recognized in bdy riemann solver");
  }

  check_cuda_error("After", __FILE__, __LINE__);
}

// wrapper for gpu kernel to calculate transformed discontinuous viscous flux at solution points
void calc_tdisvisf_upts_gpu_kernel_wrapper(int in_n_upts_per_ele, int in_n_dims, int in_n_fields, int in_n_eles, int in_ele_type, double in_filter_ratio, int LES, int SGS_model, double* in_Lu_ptr, double* in_Le_ptr, double* in_disu_upts_ptr, double* out_tdisf_upts_ptr, double* in_grad_disu_upts_ptr, double* in_detjac_upts_ptr, double* in_inv_detjac_mul_jac_upts_ptr, double in_gamma, double in_prandtl, double in_rt_inf, double in_mu_inf, double in_c_sth, double in_fix_vis, int equation, double in_diff_coeff)
{
	// HACK: fix 256 threads per block
	int n_blocks=((in_n_eles*in_n_upts_per_ele-1)/256)+1;

  check_cuda_error("Before", __FILE__, __LINE__);

  if (equation==0)
  {
    if (in_n_dims==2)
	    calc_tdisvisf_upts_NS_gpu_kernel<2,4,3> <<<n_blocks,256>>>(in_n_upts_per_ele, in_n_eles, in_ele_type, in_filter_ratio, LES, SGS_model, in_Lu_ptr, in_Le_ptr, in_disu_upts_ptr, out_tdisf_upts_ptr, in_grad_disu_upts_ptr, in_detjac_upts_ptr, in_inv_detjac_mul_jac_upts_ptr, in_gamma, in_prandtl, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis);
    else if (in_n_dims==3)
	    calc_tdisvisf_upts_NS_gpu_kernel<3,5,6> <<<n_blocks,256>>>(in_n_upts_per_ele, in_n_eles, in_ele_type, in_filter_ratio, LES, SGS_model, in_Lu_ptr, in_Le_ptr, in_disu_upts_ptr, out_tdisf_upts_ptr, in_grad_disu_upts_ptr, in_detjac_upts_ptr, in_inv_detjac_mul_jac_upts_ptr, in_gamma, in_prandtl, in_rt_inf, in_mu_inf, in_c_sth, in_fix_vis);
    else
		  FatalError("ERROR: Invalid number of dimensions ... ");
  }
  else if (equation==1)
  {
    if (in_n_dims==2)
	    calc_tdisvisf_upts_AD_gpu_kernel<2> <<<n_blocks,256>>>(in_n_upts_per_ele, in_n_eles, in_disu_upts_ptr, out_tdisf_upts_ptr, in_grad_disu_upts_ptr, in_detjac_upts_ptr, in_inv_detjac_mul_jac_upts_ptr, in_diff_coeff);
    else if (in_n_dims==3)
	    calc_tdisvisf_upts_AD_gpu_kernel<3> <<<n_blocks,256>>>(in_n_upts_per_ele, in_n_eles, in_disu_upts_ptr, out_tdisf_upts_ptr, in_grad_disu_upts_ptr, in_detjac_upts_ptr, in_inv_detjac_mul_jac_upts_ptr, in_diff_coeff);
    else
		  FatalError("ERROR: Invalid number of dimensions ... ");
  }
  else 
    FatalError("equation not recognized");

  check_cuda_error("After",__FILE__, __LINE__);
}

// wrapper for gpu kernel to transform gradient at sol points to physical gradient
void transform_grad_disu_upts_kernel_wrapper(int in_n_upts_per_ele, int in_n_dims, int in_n_fields, int in_n_eles, double* in_grad_disu_upts_ptr, double* in_detjac_upts_ptr, double* in_inv_detjac_mul_jac_upts_ptr, int equation) 
{
	// HACK: fix 256 threads per block
	int n_blocks=((in_n_eles*in_n_upts_per_ele-1)/256)+1;

  check_cuda_error("Before", __FILE__, __LINE__);

  if(equation == 0) {
    if (in_n_dims==2)
	    transform_grad_disu_upts_kernel<2,4> <<<n_blocks,256>>>(in_n_upts_per_ele,in_n_eles,in_grad_disu_upts_ptr,in_detjac_upts_ptr,in_inv_detjac_mul_jac_upts_ptr);
    else if (in_n_dims==3)
	    transform_grad_disu_upts_kernel<3,5> <<<n_blocks,256>>>(in_n_upts_per_ele,in_n_eles,in_grad_disu_upts_ptr,in_detjac_upts_ptr,in_inv_detjac_mul_jac_upts_ptr);
    else
		  FatalError("ERROR: Invalid number of dimensions ... ");
  }
  else if(equation == 1) {
    if (in_n_dims==2)
	    transform_grad_disu_upts_kernel<2,1> <<<n_blocks,256>>>(in_n_upts_per_ele,in_n_eles,in_grad_disu_upts_ptr,in_detjac_upts_ptr,in_inv_detjac_mul_jac_upts_ptr);
    else if (in_n_dims==3)
	    transform_grad_disu_upts_kernel<3,1> <<<n_blocks,256>>>(in_n_upts_per_ele,in_n_eles,in_grad_disu_upts_ptr,in_detjac_upts_ptr,in_inv_detjac_mul_jac_upts_ptr);
    else
		  FatalError("ERROR: Invalid number of dimensions ... ");
  }
  else
    FatalError("equation not recognized");

  check_cuda_error("After",__FILE__, __LINE__);
}


// wrapper for gpu kernel to calculate normal transformed continuous viscous flux at the flux points
void calc_norm_tconvisf_fpts_gpu_kernel_wrapper(int in_n_fpts_per_inter, int in_n_dims, int in_n_fields, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_grad_disu_fpts_l_ptr, double** in_grad_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_norm_tconf_fpts_r_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr, double** in_norm_fpts_ptr, int in_riemann_solve_type, int in_vis_riemann_solve_type, double in_pen_fact, double in_tau, double in_gamma, double in_prandtl, double in_rt_inf, double in_mu_inf, double in_c_sth, double in_fix_vis, int equation, double in_diff_coeff)
{
	// HACK: fix 256 threads per block
	int n_blocks=((in_n_inters*in_n_fpts_per_inter-1)/256)+1;

  check_cuda_error("Before", __FILE__, __LINE__);

  if(equation==0)
  {
    if (in_vis_riemann_solve_type==0) // LDG
    {
      if (in_n_dims==2)
        calc_norm_tconvisf_fpts_NS_gpu_kernel<2,4,3,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_grad_disu_fpts_l_ptr,in_grad_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_norm_tconf_fpts_r_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr,in_norm_fpts_ptr,in_pen_fact,in_tau,in_gamma,in_prandtl,in_rt_inf, in_mu_inf,in_c_sth,in_fix_vis);
      else if (in_n_dims==3)
        calc_norm_tconvisf_fpts_NS_gpu_kernel<3,5,6,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_grad_disu_fpts_l_ptr,in_grad_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_norm_tconf_fpts_r_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr,in_norm_fpts_ptr,in_pen_fact,in_tau,in_gamma,in_prandtl,in_rt_inf, in_mu_inf,in_c_sth,in_fix_vis);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized ... ");
  }
  else if(equation==1)
  {
    if (in_vis_riemann_solve_type==0) // LDG
    {
      if (in_n_dims==2)
        calc_norm_tconvisf_fpts_AD_gpu_kernel<2> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_grad_disu_fpts_l_ptr,in_grad_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_norm_tconf_fpts_r_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr,in_norm_fpts_ptr,in_pen_fact,in_tau,in_diff_coeff);
      else if (in_n_dims==3)
        calc_norm_tconvisf_fpts_AD_gpu_kernel<3> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_grad_disu_fpts_l_ptr,in_grad_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_norm_tconf_fpts_r_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_r_ptr,in_norm_fpts_ptr,in_pen_fact,in_tau,in_diff_coeff);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized ... ");
  }
  else 
    FatalError("equation not recognized");


  check_cuda_error("After", __FILE__, __LINE__);
}

// wrapper for gpu kernel to calculate normal transformed continuous viscous flux at the flux points at boundaries
void calc_norm_tconvisf_fpts_boundary_gpu_kernel_wrapper(int in_n_fpts_per_inter, int in_n_dims, int in_n_fields, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_grad_disu_fpts_l_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_norm_fpts_ptr, double** in_loc_fpts_ptr, int* in_boundary_type, double* in_bdy_params, double** in_delta_disu_fpts_l_ptr, int in_riemann_solve_type, int in_vis_riemann_solve_type, double in_R_ref, double in_pen_fact, double in_tau, double in_gamma, double in_prandtl, double in_rt_inf, double in_mu_inf, double in_c_sth, double in_fix_vis, double in_time_bound, int in_equation, double in_diff_coeff)
{

	// HACK: fix 256 threads per block
	int n_blocks=((in_n_inters*in_n_fpts_per_inter-1)/256)+1;

  check_cuda_error("Before", __FILE__, __LINE__);

  if (in_vis_riemann_solve_type==0) // LDG
  {
		if(in_equation==0)
		{
    	if (in_n_dims==2)
      	calc_norm_tconvisf_fpts_boundary_gpu_kernel<2,4,3,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_grad_disu_fpts_l_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_loc_fpts_ptr,in_boundary_type,in_bdy_params,in_delta_disu_fpts_l_ptr,in_R_ref,in_pen_fact,in_tau,in_gamma,in_prandtl,in_rt_inf,in_mu_inf,in_c_sth,in_fix_vis, in_time_bound, in_equation, in_diff_coeff);
    	else if (in_n_dims==3)
      	calc_norm_tconvisf_fpts_boundary_gpu_kernel<3,5,6,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_grad_disu_fpts_l_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_loc_fpts_ptr,in_boundary_type,in_bdy_params,in_delta_disu_fpts_l_ptr,in_R_ref,in_pen_fact,in_tau,in_gamma,in_prandtl,in_rt_inf,in_mu_inf,in_c_sth,in_fix_vis, in_time_bound, in_equation, in_diff_coeff);
  	}
		else if(in_equation==1)
		{
    	if (in_n_dims==2)
      	calc_norm_tconvisf_fpts_boundary_gpu_kernel<2,1,1,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_grad_disu_fpts_l_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_loc_fpts_ptr,in_boundary_type,in_bdy_params,in_delta_disu_fpts_l_ptr,in_R_ref,in_pen_fact,in_tau,in_gamma,in_prandtl,in_rt_inf,in_mu_inf,in_c_sth,in_fix_vis, in_time_bound, in_equation, in_diff_coeff);
    	else if (in_n_dims==3)
      	calc_norm_tconvisf_fpts_boundary_gpu_kernel<3,1,1,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_grad_disu_fpts_l_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_loc_fpts_ptr,in_boundary_type,in_bdy_params,in_delta_disu_fpts_l_ptr,in_R_ref,in_pen_fact,in_tau,in_gamma,in_prandtl,in_rt_inf,in_mu_inf,in_c_sth,in_fix_vis, in_time_bound, in_equation, in_diff_coeff);
		}
	}
  else
		FatalError("ERROR: Viscous riemann solver type not recognized ... ");

  check_cuda_error("After", __FILE__, __LINE__);
}


#ifdef _MPI

void pack_out_buffer_disu_gpu_kernel_wrapper(int in_n_fpts_per_inter,int in_n_inters,int in_n_fields,double** in_disu_fpts_l_ptr, double* in_out_buffer_disu_ptr)
{
  int block_size=256;
	int n_blocks=((in_n_inters*in_n_fpts_per_inter-1)/256)+1;

  check_cuda_error("Before", __FILE__, __LINE__);

  if (in_n_fields==1)
    pack_out_buffer_disu_gpu_kernel<1> <<< n_blocks,block_size >>> (in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_out_buffer_disu_ptr);
  else if (in_n_fields==4)
    pack_out_buffer_disu_gpu_kernel<4> <<< n_blocks,block_size >>> (in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_out_buffer_disu_ptr);
  else if (in_n_fields==5)
    pack_out_buffer_disu_gpu_kernel<5> <<< n_blocks,block_size >>> (in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_out_buffer_disu_ptr);
  else
    FatalError("Number of fields not supported in pack_out_buffer");

  check_cuda_error("After", __FILE__, __LINE__);

}

void pack_out_buffer_grad_disu_gpu_kernel_wrapper(int in_n_fpts_per_inter,int in_n_inters,int in_n_fields,int in_n_dims, double** in_grad_disu_fpts_l_ptr, double* in_out_buffer_grad_disu_ptr)
{
  int block_size=256;
	int n_blocks=((in_n_inters*in_n_fpts_per_inter*in_n_dims-1)/256)+1;

  check_cuda_error("Before", __FILE__, __LINE__);

  if (in_n_fields==1)
  {
    if (in_n_dims==2) {
      pack_out_buffer_grad_disu_gpu_kernel<1,2> <<< n_blocks,block_size >>> (in_n_fpts_per_inter,in_n_inters,in_grad_disu_fpts_l_ptr,in_out_buffer_grad_disu_ptr);
    }
    else if (in_n_dims==3) {
      pack_out_buffer_grad_disu_gpu_kernel<1,3> <<< n_blocks,block_size >>> (in_n_fpts_per_inter,in_n_inters,in_grad_disu_fpts_l_ptr,in_out_buffer_grad_disu_ptr);
    }

  }
  else if (in_n_fields==4)
  {
    pack_out_buffer_grad_disu_gpu_kernel<4,2> <<< n_blocks,block_size >>> (in_n_fpts_per_inter,in_n_inters,in_grad_disu_fpts_l_ptr,in_out_buffer_grad_disu_ptr);
  }
  else if (in_n_fields==5)
  {
    pack_out_buffer_grad_disu_gpu_kernel<5,3> <<< n_blocks,block_size >>> (in_n_fpts_per_inter,in_n_inters,in_grad_disu_fpts_l_ptr,in_out_buffer_grad_disu_ptr);
  }
  else
    FatalError("Number of fields not supported in pack_out_buffer");

  check_cuda_error("After", __FILE__, __LINE__);

}

// wrapper for gpu kernel to calculate normal transformed continuous inviscid flux at the flux points
void calc_norm_tconinvf_fpts_mpi_gpu_kernel_wrapper(int in_n_fpts_per_inter, int in_n_dims, int in_n_fields, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_norm_fpts_ptr,int in_riemann_solve_type, double** in_delta_disu_fpts_l_ptr, double in_gamma, double in_pen_fact,  int in_viscous, int in_vis_riemann_solve_type, double wave_speed_x, double wave_speed_y, double wave_speed_z, double lambda)
{
  
  int block_size=256;
	int n_blocks=((in_n_inters*in_n_fpts_per_inter-1)/block_size)+1;

  check_cuda_error("Before", __FILE__, __LINE__);

  if (in_riemann_solve_type==0 ) // Rusanov
  {
    if (in_vis_riemann_solve_type==0 ) //LDG
    {
      if (in_n_dims==2)
	      calc_norm_tconinvf_fpts_NS_mpi_gpu_kernel<2,4,0,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_gamma,in_pen_fact,in_viscous);
      else if (in_n_dims==3)
	      calc_norm_tconinvf_fpts_NS_mpi_gpu_kernel<3,5,0,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_gamma,in_pen_fact,in_viscous);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized ... ");
  }
  else if (in_riemann_solve_type==2 ) // Roe
  {
    if (in_vis_riemann_solve_type==0 ) //LDG
    {
      if (in_n_dims==2)
	      calc_norm_tconinvf_fpts_NS_mpi_gpu_kernel<2,4,2,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_gamma,in_pen_fact,in_viscous);
      else if (in_n_dims==3)
	      calc_norm_tconinvf_fpts_NS_mpi_gpu_kernel<3,5,2,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_gamma,in_pen_fact,in_viscous);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized ... ");
  }
  else if (in_riemann_solve_type==1) // Lax-Friedrich
  {
    if(in_vis_riemann_solve_type==0) //LDG
    {
      if (in_n_dims==2)
	      calc_norm_tconinvf_fpts_lax_friedrich_mpi_gpu_kernel<2,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_pen_fact,in_viscous,wave_speed_x,wave_speed_y,wave_speed_z,lambda);
      else if (in_n_dims==3)
	      calc_norm_tconinvf_fpts_lax_friedrich_mpi_gpu_kernel<3,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_delta_disu_fpts_l_ptr,in_pen_fact,in_viscous,wave_speed_x,wave_speed_y,wave_speed_z,lambda);
    }
    else
		  FatalError("ERROR: Viscous riemann solver type not recognized ... ");
  }
  else
  {
		FatalError("ERROR: Riemann solver type not recognized ... ");
  }

  check_cuda_error("After", __FILE__, __LINE__);

}


// wrapper for gpu kernel to calculate normal transformed continuous viscous flux at the flux points
void calc_norm_tconvisf_fpts_mpi_gpu_kernel_wrapper(int in_n_fpts_per_inter, int in_n_dims, int in_n_fields, int in_n_inters, double** in_disu_fpts_l_ptr, double** in_disu_fpts_r_ptr, double** in_grad_disu_fpts_l_ptr, double** in_grad_disu_fpts_r_ptr, double** in_norm_tconf_fpts_l_ptr, double** in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr, double** in_norm_fpts_ptr, int in_riemann_solve_type, int in_vis_riemann_solve_type, double in_pen_fact, double in_tau, double in_gamma, double in_prandtl, double in_rt_inf, double in_mu_inf, double in_c_sth, double in_fix_vis, double in_diff_coeff)
{
 	// HACK: fix 256 threads per block
	int n_blocks=((in_n_inters*in_n_fpts_per_inter-1)/256)+1;

  check_cuda_error("Before", __FILE__, __LINE__);
  
	if (in_riemann_solve_type==0 ) // Rusanov
  {
  	if (in_vis_riemann_solve_type==0) // LDG
  	{
  	  if (in_n_dims==2)
  	    calc_norm_tconvisf_fpts_NS_mpi_gpu_kernel<2,4,3,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_grad_disu_fpts_l_ptr,in_grad_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_pen_fact,in_tau,in_gamma,in_prandtl,in_rt_inf, in_mu_inf,in_c_sth,in_fix_vis);
  	  else if (in_n_dims==3)
  	    calc_norm_tconvisf_fpts_NS_mpi_gpu_kernel<3,5,6,0> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_grad_disu_fpts_l_ptr,in_grad_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_pen_fact,in_tau,in_gamma,in_prandtl,in_rt_inf, in_mu_inf,in_c_sth,in_fix_vis);
  	}
  	else
			FatalError("ERROR: Viscous riemann solver type not recognized ... ");
	}
  else if (in_riemann_solve_type==1) // Lax-Friedrich
  {
    if (in_vis_riemann_solve_type==0) // LDG
    {
      if (in_n_dims==2)
        calc_norm_tconvisf_fpts_AD_mpi_gpu_kernel<2> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_grad_disu_fpts_l_ptr,in_grad_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_pen_fact,in_tau,in_diff_coeff);
      else if (in_n_dims==3)
        calc_norm_tconvisf_fpts_AD_mpi_gpu_kernel<3> <<<n_blocks,256>>>(in_n_fpts_per_inter,in_n_inters,in_disu_fpts_l_ptr,in_disu_fpts_r_ptr,in_grad_disu_fpts_l_ptr,in_grad_disu_fpts_r_ptr,in_norm_tconf_fpts_l_ptr,in_mag_tnorm_dot_inv_detjac_mul_jac_fpts_l_ptr,in_norm_fpts_ptr,in_pen_fact,in_tau,in_diff_coeff);
    }
  	else
			FatalError("ERROR: Viscous riemann solver type not recognized ... ");
	}
  else
  {
		FatalError("ERROR: Riemann solver type not recognized ... ");
  }

  check_cuda_error("After", __FILE__, __LINE__);

}

#endif

void bespoke_SPMV(int m, int n, int n_fields, int n_eles, double* opp_ell_data_ptr, int* opp_ell_indices_ptr, int nnz_per_row, double* b_ptr, double *c_ptr, int cell_type, int order, int add_flag)
{

  int eles_per_block=2; // allows up to 128 DOFs per element
  int grid_size = (n_eles-1)/(eles_per_block)+1; 
  int block_size = eles_per_block*m;
  int shared_mem = n*eles_per_block*n_fields;
  shared_mem += shared_mem/HALFWARP;

  if (n_fields==1)
  {
    bespoke_SPMV_kernel<1> <<<grid_size, block_size, shared_mem*sizeof(double) >>> (c_ptr, b_ptr, opp_ell_data_ptr, opp_ell_indices_ptr, nnz_per_row, n_eles, n, m, eles_per_block,n_eles*n,n_eles*m,add_flag);
  }
  else if (n_fields==4)
  {
    bespoke_SPMV_kernel<4> <<<grid_size, block_size, shared_mem*sizeof(double) >>> (c_ptr, b_ptr, opp_ell_data_ptr, opp_ell_indices_ptr, nnz_per_row, n_eles, n, m, eles_per_block,n_eles*n,n_eles*m,add_flag);
  }
  else if (n_fields==5)
  {
    bespoke_SPMV_kernel<5> <<<grid_size, block_size, shared_mem*sizeof(double) >>> (c_ptr, b_ptr, opp_ell_data_ptr, opp_ell_indices_ptr, nnz_per_row, n_eles, n, m, eles_per_block,n_eles*n,n_eles*m,add_flag);
  }

}


